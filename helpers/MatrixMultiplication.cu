#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "helpers/MatrixMultiplication.cuh"
#include <nvfunctional>


using CudaFunction = nvstd::function<double*(double*, int)>;

// CUDA kernel for matrix-vector multiplication
__global__ void MatrixMultiplication(double* d_matrix, double* d_vectors, double* d_result, int rows, int cols, int vec_num) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < vec_num) {
        double sum = 0.0;
        for (int i = 0; i < cols; ++i) {
            sum += d_matrix[row * cols + i] * d_vectors[i * vec_num + col];
        }
        d_result[row * vec_num + col] = sum;
    }
}

// __global__ void Sigma_function(double* d_sigmapts, double* d_pts, double* input_vector, double* pts_vector, int sigmapts_rows, int sigmapts_cols, int res_rows, int res_cols, FunctionPtr func_ptr, void* context){

__global__ void Sigma_function(double* d_sigmapts, double* d_pts, double* mu, int sigmapts_rows, int sigmapts_cols, int res_rows, int res_cols, FunctionPtr func_ptr, void* context, int type){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double function_value = cost_function1(d_sigmapts, sigmapts_cols);

    if (idx < sigmapts_rows){
        if (type == 0) //res.size = 1*1
            d_pts[idx] = function_value;
        else if (type == 1){
            // printf("Here");
            for (int i=0; i<sigmapts_cols; i++)
                d_pts[idx*sigmapts_cols + i] = (d_sigmapts[idx*sigmapts_cols + i] - mu[idx*sigmapts_cols + i]) * function_value;
            // dim3 blockSize1(3);
            // dim3 threadperblock1((sigmapts_cols + blockSize1.x - 1) / blockSize1.x);
            // func_Vmu<<<blockSize1, threadperblock1>>>(d_sigmapts, d_pts, mu, sigmapts_cols, idx, function_value);
        }
        else{
            for (int i=0; i<sigmapts_cols; i++)
                for (int j=0; j<sigmapts_cols; j++)
                    d_pts[idx*sigmapts_cols *sigmapts_cols+ i*sigmapts_cols +j] = (d_sigmapts[idx*sigmapts_cols + i] - mu[idx*sigmapts_cols + i]) * (d_sigmapts[idx*sigmapts_cols + j] - mu[idx*sigmapts_cols + j]) * function_value;

            // dim3 blockSize2(3, 3);
            // dim3 threadperblock2((sigmapts_cols + blockSize2.x - 1) / blockSize2.x, (sigmapts_cols + blockSize2.y - 1) / blockSize2.y);
            // func_Vmu<<<blockSize2, threadperblock2>>>(d_sigmapts + idx*sigmapts_cols, d_pts + idx*res_rows*res_cols, mu, sigmapts_cols, function_value);
        }
        hipDeviceSynchronize();
    }
}

__global__ void obtain_res(double* d_pts, double* d_weights, double* d_result, int sigmapts_rows, int res_rows, int res_cols){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < res_rows && col < res_cols){
        double sum = 0;
        for(int i = 0; i < sigmapts_rows; i++){
            sum += d_pts[i*res_rows*res_cols + row*res_cols + col] * d_weights[i];
        }
        d_result[row*res_cols + col] = sum;
    }
    // printf("sigma:%lf \n",d_result[0]);
}

__global__ void func_Vmu(const double* vec_x, double* pt, const double* mu, int dim, int index, double& func_value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < dim){
        pt[index*dim + idx] = (vec_x[index*dim + idx] - mu[idx]) * func_value;
    }
}

__global__ void func_Vmumu(const double* vec_x, double* pt, const double* mu, int dim, int index, double& func_value) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < dim && col < dim) {
        pt[row * dim + col] = (vec_x[row] - mu[row]) * (vec_x[col] - mu[col]) * func_value;
    }
}

// __device__ void calculate_derivative(const double* vec_x, double* pt, const double* mu, int dim, double& func_value, int& type){
//     dim3 blockSize(3, 3);
//     dim3 threadperblock((dim + blockSize.x - 1) / blockSize.x, (dim + blockSize.y - 1) / blockSize.y);
//     if (type == 1)

// }

__host__ __device__ double cost_function1(const double* vec_x, int dim) {
    double x = vec_x[0];
    double mu_p = 20, f = 400, b = 0.1, sig_r_sq = 0.09;
    double sig_p_sq = 9;

    // y should be sampled. for single trial just give it a value.
    double y = f*b/mu_p - 0.8;

    return ((x - mu_p)*(x - mu_p) / sig_p_sq / 2 + (y - f*b/x)*(y - f*b/x) / sig_r_sq / 2); 
}



void MatrixMul(double* matrix, double* vectorMatrix, double* result, int rows, int cols, int vec_num)
{   
    double *matrix_gpu, *vectorMatrix_gpu, *result_gpu;  

    hipMalloc(&matrix_gpu, rows * cols * sizeof(double));
    hipMalloc(&vectorMatrix_gpu, cols * vec_num * sizeof(double));
    hipMalloc(&result_gpu, rows * vec_num * sizeof(double));

    // Copy the data from host to device
    hipMemcpy(matrix_gpu, matrix, rows * cols * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(vectorMatrix_gpu, vectorMatrix, cols * vec_num * sizeof(double), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 blockSize(4, 4);
    dim3 threadperblock((vec_num + blockSize.x - 1) / blockSize.x, (rows + blockSize.y - 1) / blockSize.y);

        
    MatrixMultiplication<<<threadperblock, blockSize>>>(matrix_gpu, vectorMatrix_gpu, result_gpu, rows, cols, vec_num);
    hipDeviceSynchronize();
    hipMemcpy(result, result_gpu, rows * vec_num * sizeof(double), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(matrix_gpu);
    hipFree(vectorMatrix_gpu);
    hipFree(result_gpu);
}

// void CudaIntegration(FunctionPtr func_ptr, double* d_sigmapts, double* d_weights, double* d_results, int sigma_rows, int sigma_cols, int res_rows, int res_cols, void* context)
void CudaIntegration(FunctionPtr function, double* d_sigmapts, double* d_weights, double* d_results, double* d_mu, int sigma_rows, int sigma_cols, int res_rows, int res_cols, void* context, double* d_pts1, double* d_pts2, int type)
{
    double *sigmapts_gpu, *pts_gpu, *weight_gpu, *result_gpu, *mu_gpu; 

    hipMalloc(&sigmapts_gpu, sigma_rows * sigma_cols * sizeof(double));
    hipMalloc(&pts_gpu, sigma_rows * res_rows * res_cols * sizeof(double));
    hipMalloc(&weight_gpu, sigma_rows * sizeof(double));
    hipMalloc(&result_gpu, res_rows * res_cols * sizeof(double));
    hipMalloc(&mu_gpu, sigma_cols * sizeof(double));

    hipMemcpy(sigmapts_gpu, d_sigmapts, sigma_rows * sigma_cols * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(weight_gpu, d_weights, sigma_rows * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(mu_gpu, d_mu, sigma_cols * sizeof(double), hipMemcpyHostToDevice);

    // Dimension for the first kernel function
    dim3 blockSize1(3);
    dim3 threadperblock1((sigma_rows + blockSize1.x - 1) / blockSize1.x);

    // Kernel 1: Obtain the result of function 
    // Sigma_function<<<threadperblock1, blockSize1>>>(sigmapts_gpu, pts_gpu, input_vector, pts_vector, sigma_rows, sigma_cols, res_rows, res_cols, func_ptr, context);

    Sigma_function<<<blockSize1, threadperblock1>>>(sigmapts_gpu, pts_gpu, mu_gpu, sigma_rows, sigma_cols, res_rows, res_cols, function, context, type);
    hipDeviceSynchronize();
    hipMemcpy(d_pts2, pts_gpu, sigma_rows * res_rows * res_cols * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(pts_gpu, d_pts1, sigma_rows * res_rows * res_cols * sizeof(double), hipMemcpyHostToDevice);
    
    // Dimension for the second kernel function
    dim3 blockSize2(3, 3);
    dim3 threadperblock2((res_cols + blockSize2.x - 1) / blockSize2.x, (res_rows + blockSize2.y - 1) / blockSize2.y);

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    obtain_res<<<blockSize2, threadperblock2>>>(pts_gpu, weight_gpu, result_gpu, sigma_rows, res_rows, res_cols);
    hipDeviceSynchronize();
    hipMemcpy(d_results, result_gpu, res_rows * res_cols * sizeof(double), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(sigmapts_gpu);
    hipFree(pts_gpu);
    hipFree(weight_gpu);
    hipFree(result_gpu);
    
}


void CudaIntegration1(double* d_pts, double* d_weights, double* d_results, int sigma_rows, int sigma_cols, int res_rows, int res_cols)
{
    double *pts_gpu, *weight_gpu, *result_gpu; 

    hipMalloc(&pts_gpu, sigma_rows * res_rows * res_cols * sizeof(double));
    hipMalloc(&weight_gpu, sigma_rows * sizeof(double));
    hipMalloc(&result_gpu, res_rows * res_cols * sizeof(double));

    hipMemcpy(pts_gpu, d_pts, sigma_rows * res_rows * res_cols * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(weight_gpu, d_weights, sigma_rows * sizeof(double), hipMemcpyHostToDevice);

    // Dimension for the second kernel function
    dim3 blockSize(3, 3);
    dim3 threadperblock((res_cols + blockSize.x - 1) / blockSize.x, (res_rows + blockSize.y - 1) / blockSize.y);

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    obtain_res<<<blockSize, threadperblock>>>(pts_gpu, weight_gpu, result_gpu, sigma_rows, res_rows, res_cols);
    hipDeviceSynchronize();
    hipMemcpy(d_results, result_gpu, res_rows * res_cols * sizeof(double), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(pts_gpu);
    hipFree(weight_gpu);
    hipFree(result_gpu);

}
