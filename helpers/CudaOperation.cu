#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "helpers/CudaOperation.h"
#include "helpers/timer.h"

using namespace Eigen;

template class CudaOperation_Base<PlanarSDF>;
template class CudaOperation_Base<SignedDistanceField>;

void printGPUMemoryInfo() {
    size_t free_mem = 0, total_mem = 0;
    hipError_t err = hipMemGetInfo(&free_mem, &total_mem);
    if (err != hipSuccess) {
        std::cerr << "hipMemGetInfo failed: " << hipGetErrorString(err) << std::endl;
        return;
    }
    std::cout << "GPU Memory: free = " << free_mem / (1024.0 * 1024.0) << " MB" << std::endl;
}


template <typename RobotType>
__global__ void Sigma_function(double* d_sigmapts, double* d_pts, double* mu,
                               int sigmapts_rows, int sigmapts_cols, int res_rows, int res_cols, int type, 
                               RobotType* pointer, double* d_data){
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < res_rows && col < res_cols*sigmapts_rows){
        int idx = col / res_cols;
        Eigen::Map<MatrixXd> sigmapts(d_sigmapts, sigmapts_rows, sigmapts_cols);

        (pointer->_sdf).data_array_ = d_data;

        double function_value = pointer -> cost_obstacle_planar(sigmapts.row(idx), pointer->_sdf);

        if (type == 0)
            d_pts[idx*res_rows + row] = function_value;
        else if (type == 1)
            d_pts[idx*res_rows + row] = (d_sigmapts[idx + sigmapts_rows * row] - mu[row]) * function_value;
        else{
            int r = col % res_cols;
            d_pts[idx*sigmapts_cols*sigmapts_cols+ r*sigmapts_cols + row] = (d_sigmapts[idx + sigmapts_rows * row] - mu[row]) * (d_sigmapts[idx + sigmapts_rows * r] - mu[r]) * function_value;
        }
    }
}

__global__ void Sigma_function(double* d_sigmapts, double* d_pts, double* mu,
                               int sigmapts_rows, int sigmapts_cols, int res_rows, int res_cols, int type, 
                               gvi::CudaOperation_3dArm* pointer, double* d_data){
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < res_rows && col < res_cols*sigmapts_rows){
        int idx = col / res_cols;
        Eigen::Map<MatrixXd> sigmapts(d_sigmapts, sigmapts_rows, sigmapts_cols);

        (pointer->_sdf).data_array_ = d_data;

        double function_value = pointer -> cost_obstacle(sigmapts.row(idx), pointer->_sdf, pointer->_fk);

        if (type == 0)
            d_pts[idx*res_rows + row] = function_value;
        else if (type == 1)
            d_pts[idx*res_rows + row] = (d_sigmapts[idx + sigmapts_rows * row] - mu[row]) * function_value;
        else{
            int r = col % res_cols;
            d_pts[idx*sigmapts_cols*sigmapts_cols+ r*sigmapts_cols + row] = (d_sigmapts[idx + sigmapts_rows * row] - mu[row]) * (d_sigmapts[idx + sigmapts_rows * r] - mu[r]) * function_value;
        }
    }
}

__global__ void obtain_res(double* d_pts, double* d_weights, double* d_result, int sigmapts_rows, int res_rows, int res_cols){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < res_rows && col < res_cols){
        double sum = 0;
        for(int i = 0; i < sigmapts_rows; i++){
            sum += d_pts[i*res_rows*res_cols + col*res_rows + row] * d_weights[i];
        }
        d_result[col*res_rows + row] = sum;
    }
}

template <typename RobotType>
__global__ void cost_function(double* d_sigmapts, double* d_pts, int sigmapts_rows, int sigmapts_cols, 
                                int n_states, RobotType* pointer, double* d_data){
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < sigmapts_rows && col < n_states){
        Eigen::Map<MatrixXd> sigmapts(d_sigmapts + col*sigmapts_rows*sigmapts_cols, sigmapts_rows, sigmapts_cols);

        (pointer->_sdf).data_array_ = d_data;

        double function_value = pointer -> cost_obstacle_planar(sigmapts.row(row), pointer->_sdf);

        d_pts[col*sigmapts_rows + row] = function_value;
    }
}


__global__ void cost_function(double* d_sigmapts, double* d_pts, int sigmapts_rows, int sigmapts_cols, 
                                int n_states, gvi::CudaOperation_3dArm* pointer, double* sdf_data, 
                                double* a_data, double* alpha_data, double* d_data, double* theta_data,
                                double* rad_data, int* frames_data, double* centers_data){
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < sigmapts_rows && col < n_states){
        Eigen::Map<MatrixXd> sigmapts(d_sigmapts + col*sigmapts_rows*sigmapts_cols, sigmapts_rows, sigmapts_cols);

        (pointer->_sdf).data_array_ = sdf_data;

        (pointer->_fk)._a_data = a_data;
        (pointer->_fk)._alpha_data = alpha_data;
        (pointer->_fk)._d_data = d_data;
        (pointer->_fk)._theta_bias_data = theta_data;

        pointer->_radii_data = rad_data;
        (pointer->_fk)._frames_data = frames_data;
        (pointer->_fk)._centers_data = centers_data;

        double function_value = pointer -> cost_obstacle(sigmapts.row(row), pointer->_sdf, pointer->_fk);

        d_pts[col*sigmapts_rows + row] = function_value;
    }
}

__global__ void dmu_function(double* d_sigmapts, double* d_mu, double* d_pts, double* d_vec, int sigmapts_rows, int dim_conf, int n_states){
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < sigmapts_rows && col < n_states*dim_conf){
        int idx = col / dim_conf;
        int r = col % dim_conf;

        d_vec[col*sigmapts_rows + row] = (d_sigmapts[col*sigmapts_rows + row] - d_mu[idx*dim_conf + r]) * d_pts[idx * sigmapts_rows + row];
    }
}

__global__ void ddmu_function(double* d_sigmapts, double* d_mu, double* d_pts, double* d_vec, int sigmapts_rows, int sigmapts_cols, int n_states){
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < sigmapts_rows*sigmapts_cols && col < n_states*sigmapts_cols){
        int idx_x = col / sigmapts_cols; // Use which mu
        int idx_y = row / sigmapts_cols; // use which sigma
        int mat_x = col % sigmapts_cols; // 
        int mat_y = row % sigmapts_cols;

        d_vec[col*sigmapts_rows*sigmapts_cols + row] = (d_sigmapts[(idx_x*sigmapts_cols + mat_x) * sigmapts_rows + idx_y] - d_mu[idx_x*sigmapts_cols + mat_x]) 
                                                     * (d_sigmapts[(idx_x*sigmapts_cols + mat_y) * sigmapts_rows + idx_y] - d_mu[idx_x*sigmapts_cols + mat_y]) 
                                                     * d_pts[idx_x * sigmapts_rows + idx_y];
    }
}

__global__ void obtain_cost(double* d_pts, double* d_weights, double* d_result, int sigmapts_rows, int n_states){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n_states){
        double sum = 0;
        for(int i = 0; i < sigmapts_rows; i++){
            sum += d_pts[idx*sigmapts_rows + i]* d_weights[i];
        }
        d_result[idx] = sum;
    }
}

__global__ void obtain_dmu(double* d_vec, double* d_weights, double* d_result, int sigmapts_rows, int res_cols){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < res_cols){
        double sum = 0;
        for(int i = 0; i < sigmapts_rows; i++){
            sum += d_vec[idx*sigmapts_rows + i]* d_weights[i];
        }
        d_result[idx] = sum;
    }
}

__global__ void obtain_ddmu(double* d_vec, double* d_weights, double* d_result, int sigmapts_rows, int res_rows, int res_cols){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < res_rows && col < res_cols){
        double sum = 0;
        for(int i = 0; i < sigmapts_rows; i++){
            sum += d_vec[col*sigmapts_rows*res_rows + row + i*res_rows] * d_weights[i];
        }
        d_result[col*res_rows + row] = sum;
    }
}


__global__ void sqrtKernel(double* d_vals, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        d_vals[idx] = sqrt(d_vals[idx]);
    }
}

__global__ void addMeanKernel(double* sigmaPts, const double* mean, int num_rows, int dim_state)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = num_rows * dim_state;
    if (idx < total)
    {
        int j = idx / num_rows;
        sigmaPts[idx] += mean[j];
    }
}

__global__ void addMeanKernel_batched(double* d_sigmapts, const double* d_mean, int num_rows, int dim, int batch) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = batch * num_rows * dim;
    if (idx < total) {
    int state = idx / (num_rows * dim);
    int col   = (idx % (num_rows * dim)) / num_rows;
    d_sigmapts[idx] += d_mean[state * dim + col];
    }
}


// Combined kernel to compute the square root of eigenvalues and scale eigenvectors accordingly.
// Assumes matrices are stored in column-major order.
__global__ void sqrtAndScaleEigenvectorsKernel(const double* d_eigenvalues, const double* d_eigvec,
                                               double* d_scaledEigvec, int dim, int batch) {
    // Each block handles one matrix in the batch.
    int batch_idx = blockIdx.x;
    if (batch_idx >= batch) return;

    // Pointers for current batch's eigenvalues and eigenvector matrix.
    const double* eigenvals = d_eigenvalues + batch_idx * dim;
    const double* eigvec    = d_eigvec + batch_idx * dim * dim;
    double* scaledEigvec    = d_scaledEigvec + batch_idx * dim * dim;

    // Allocate shared memory to store sqrt(eigenvalues) for the current matrix.
    extern __shared__ double s_sqrtEig[];

    // Each thread computes part of the square root.
    for (int i = threadIdx.x; i < dim; i += blockDim.x) {
    s_sqrtEig[i] = sqrt(eigenvals[i]);
    }
    __syncthreads();

    // Scale eigenvector matrix columns by corresponding sqrt eigenvalues.
    int totalElements = dim * dim;
    for (int idx = threadIdx.x; idx < totalElements; idx += blockDim.x) {
    int col = idx / dim;  // In column-major, column index = idx / dim.
    scaledEigvec[idx] = eigvec[idx] * s_sqrtEig[col];
    }
}


namespace gvi{

// template <typename SDFType>
// void CudaOperation_Base<SDFType>::update_sigmapts(const MatrixXd& covariance, const MatrixXd& mean, int dim_conf, int num_states, MatrixXd& sigmapts){
//     // Compute the Cholesky decomposition of the covariance matrix
//     double *covariance_gpu, *mean_gpu, *d_sigmapts;
//     hipMalloc(&covariance_gpu, dim_conf * dim_conf * num_states * sizeof(double));
//     hipMalloc(&d_sigmapts, _sigmapts_rows * dim_conf * num_states * sizeof(double));
//     hipMalloc(&mean_gpu, mean.size() * sizeof(double));

//     hipMemcpy(covariance_gpu, covariance.data(), dim_conf * dim_conf * num_states * sizeof(double), hipMemcpyHostToDevice);
//     hipMemcpy(mean_gpu, mean.data(), mean.size() * sizeof(double), hipMemcpyHostToDevice);

//     const int num_streams = 20;

//     // Create a CUDA stream, cuSOLVER, and cuBLAS handle for each state
//     std::vector<hipStream_t> streams(num_streams);
//     std::vector<hipsolverHandle_t> cusolver_handles(num_streams);
//     std::vector<hipblasHandle_t> cublas_handles(num_streams);

//     // Containers to store temporary device memory for each state
//     std::vector<double*> d_eigen_values_vec(num_states, nullptr);
//     std::vector<int*>    d_info_vec(num_states, nullptr);
//     std::vector<double*> d_work_vec(num_states, nullptr);
//     std::vector<int>     Lwork_vec(num_states, 0);
//     std::vector<double*> d_V_scaled_vec(num_states, nullptr);
//     std::vector<double*> d_sqrtP_vec(num_states, nullptr);

//     const double alpha = 1.0, beta = 0.0;

//     std::cout << "Beginning: ";
//     printGPUMemoryInfo();

//     for (int i = 0; i < num_streams; i++) {
//         hipStreamCreate(&streams[i]);
//         hipsolverDnCreate(&cusolver_handles[i]);
//         hipsolverSetStream(cusolver_handles[i], streams[i]);
//         hipblasCreate(&cublas_handles[i]);
//         hipblasSetStream(cublas_handles[i], streams[i]);
//     }

//     std::cout << "Finished creating streams and handles: ";
//     printGPUMemoryInfo();

//     // Submit tasks for each state to its corresponding stream
//     for (int state = 0; state < num_states; state++) {        
//         // Each state's covariance matrix is stored at an offset in covariance_gpu
//         double* d_Pi = covariance_gpu + state * dim_conf * dim_conf;
//         // Assuming mean data occupies two rows per state; adjust offset as necessary
//         double* d_mi = mean_gpu + 2 * (state + 1) * dim_conf;
        
//         // 3.1 Allocate temporary memory for eigenvalues and info
//         hipMalloc(&d_eigen_values_vec[state], dim_conf * sizeof(double));
//         hipMalloc(&d_info_vec[state], sizeof(int));

//         int stream_idx = state % num_streams;  // Assign to one of the 10 streams
        
//         // Query workspace size required for eigen decomposition
//         hipsolverDnDsyevd_bufferSize(cusolver_handles[stream_idx],
//                                     HIPSOLVER_EIG_MODE_VECTOR,
//                                     HIPBLAS_FILL_MODE_LOWER,
//                                     dim_conf,
//                                     d_Pi,
//                                     dim_conf,
//                                     d_eigen_values_vec[state],
//                                     &Lwork_vec[state]);
//         hipMalloc(&d_work_vec[state], Lwork_vec[state] * sizeof(double));
        
//         // 3.2 Perform symmetric eigenvalue decomposition on d_Pi (d_Pi will store eigenvectors after the operation)
//         hipsolverDnDsyevd(cusolver_handles[stream_idx],
//                          HIPSOLVER_EIG_MODE_VECTOR,
//                          HIPBLAS_FILL_MODE_LOWER,
//                          dim_conf,
//                          d_Pi,
//                          dim_conf,
//                          d_eigen_values_vec[state],
//                          d_work_vec[state],
//                          Lwork_vec[state],
//                          d_info_vec[state]);
//         // hipsolverDnDsyevjBatched
        
//         int h_info;
//         hipMemcpy(&h_info, d_info_vec[state], sizeof(int), hipMemcpyDeviceToHost);
//         if (h_info != 0) {
//             std::cerr << "Eigen decomposition failed for state " << state << " with info " << h_info << std::endl;
//         }
        
//         // 3.3 Apply square root to the eigenvalues using a custom kernel (executed on the corresponding stream)
//         int threadsPerBlock = 16;
//         int blocks = (dim_conf + threadsPerBlock - 1) / threadsPerBlock;
//         sqrtKernel<<<blocks, threadsPerBlock, 0, streams[stream_idx]>>>(d_eigen_values_vec[state], dim_conf);

//         // 3.4 Use hipblasDdgmm to scale the eigenvector matrix by the square-rooted eigenvalues
//         hipMalloc(&d_V_scaled_vec[state], dim_conf * dim_conf * sizeof(double));
//         hipblasStatus_t stat = hipblasDdgmm(cublas_handles[stream_idx],
//                                           HIPBLAS_SIDE_RIGHT,
//                                           dim_conf,
//                                           dim_conf,
//                                           d_Pi,
//                                           dim_conf,
//                                           d_eigen_values_vec[state],
//                                           1,
//                                           d_V_scaled_vec[state],
//                                           dim_conf);
//         if (stat != HIPBLAS_STATUS_SUCCESS) {
//             std::cerr << "hipblasDdgmm failed for state " << state << ": " << stat << std::endl;
//         }
        
//         // 3.5 Compute sqrtP = d_V_scaled * (d_Pi)^T
//         hipMalloc(&d_sqrtP_vec[state], dim_conf * dim_conf * sizeof(double));
//         stat = hipblasDgemm(cublas_handles[stream_idx],
//                            HIPBLAS_OP_N,
//                            HIPBLAS_OP_T,
//                            dim_conf,
//                            dim_conf,
//                            dim_conf,
//                            &alpha,
//                            d_V_scaled_vec[state],
//                            dim_conf,
//                            d_Pi,
//                            dim_conf,
//                            &beta,
//                            d_sqrtP_vec[state],
//                            dim_conf);
//         if (stat != HIPBLAS_STATUS_SUCCESS) {
//             std::cerr << "hipblasDgemm for sqrtP failed for state " << state << ": " << stat << std::endl;
//         }
        
//         // 3.6 Compute the current state's portion of sigmapts.
//         // d_sigmapts is allocated contiguously on the device; each state occupies a block of _sigmapts_rows x dim_conf.
//         double* d_sigmapts_state = d_sigmapts + state * _sigmapts_rows * dim_conf;
//         stat = hipblasDgemm(cublas_handles[stream_idx], HIPBLAS_OP_N, HIPBLAS_OP_T, _sigmapts_rows, dim_conf, dim_conf, &alpha, _zeromean_gpu, _sigmapts_rows, d_sqrtP_vec[state], dim_conf, &beta, d_sigmapts_state, _sigmapts_rows);
//         if (stat != HIPBLAS_STATUS_SUCCESS) {
//             std::cerr << "hipblasDgemm for sigmapts failed for state " << state << ": " << stat << std::endl;
//         }
        
//         // 3.7 Add the mean to sigmapts (for the current state's block)
//         threadsPerBlock = 256;
//         blocks = (_sigmapts_rows * dim_conf + threadsPerBlock - 1) / threadsPerBlock;
//         addMeanKernel<<<blocks, threadsPerBlock, 0, streams[stream_idx]>>>(d_sigmapts_state, d_mi, _sigmapts_rows, dim_conf);
//     }

//     std::cout << "Finished submitting tasks to streams: ";
//     printGPUMemoryInfo();

//     for (int i = 0; i < num_streams; i++) {
//         hipStreamSynchronize(streams[i]);
//     }

//     // 5. Copy all states' sigmapts from device to host.
//     // The output matrix sigmapts should have dimensions _sigmapts_rows x (dim_conf * num_states)
//     MatrixXd sigma(_sigmapts_rows, dim_conf * num_states);
//     hipMemcpy(sigma.data(), d_sigmapts, _sigmapts_rows * dim_conf * num_states * sizeof(double), hipMemcpyDeviceToHost);
//     sigmapts = sigma;
    
//     // 6. Free temporary memory for each state and destroy handles and streams
//     for (int state = 0; state < num_states; state++) {
//         hipFree(d_eigen_values_vec[state]);
//         hipFree(d_info_vec[state]);
//         hipFree(d_work_vec[state]);
//         hipFree(d_V_scaled_vec[state]);
//         hipFree(d_sqrtP_vec[state]);
        
//         // hipsolverDnDestroy(cusolver_handles[state]);
//         // // hipblasDestroy(cublas_handles[state]);
//         // hipStreamDestroy(streams[state]);
//     }

//     for (int i = 0; i < num_streams; i++) {
//         hipsolverDnDestroy(cusolver_handles[i]);
//         // hipblasDestroy(cublas_handles[i]);
//         hipStreamDestroy(streams[i]);
//     }

//     std::cout << "Finished freeing temporary memory: ";
//     printGPUMemoryInfo();
    
//     // Free global data
//     hipFree(covariance_gpu);
//     hipFree(mean_gpu);
//     hipFree(d_sigmapts);
// }


template <typename SDFType>
void CudaOperation_Base<SDFType>::update_sigmapts(const MatrixXd& covariance, const MatrixXd& mean,
                                                  int dim_conf, int num_states, MatrixXd& sigmapts) {
    Timer timer;
    timer.start();
    // 1. Allocate device memory for covariance, mean, and sigmapts.
    double alpha = 1.0, beta = 0.0;
    double *d_covariance, *d_mean, *d_sigmapts;
    size_t covarianceSize = num_states * dim_conf * dim_conf * sizeof(double);
    size_t meanSize       = num_states * dim_conf * sizeof(double);
    size_t sigmaptsSize   = _sigmapts_rows * dim_conf * num_states * sizeof(double);

    hipMalloc(&d_covariance, covarianceSize);
    hipMalloc(&d_mean, meanSize);
    hipMalloc(&d_sigmapts, sigmaptsSize);
    
    hipMemcpy(d_covariance, covariance.data(), covarianceSize, hipMemcpyHostToDevice);
    hipMemcpy(d_mean, mean.data(), meanSize, hipMemcpyHostToDevice);

    std::cout << "Time to allocate memory: " << timer.end_mus_output() << " us" << std::endl;

    // // 2. Setup cusolver handle and create syevj parameter object.
    // timer.start();
    // hipsolverHandle_t cusolverH = nullptr;
    // hipsolverDnCreate(&cusolverH);

    // hipblasHandle_t cublasH = nullptr;
    // hipblasCreate(&cublasH);
    
    // hipsolverSyevjInfo_t syevj_params = nullptr;
    // hipsolverDnCreateSyevjInfo(&syevj_params);
    // std::cout << "Time to create handles: " << timer.end_mus_output() << " us" << std::endl;

    // 3. Allocate memory for eigenvalues and info.
    timer.start();
    double* d_eigenvalues = nullptr;
    size_t eigenvaluesSize = dim_conf * num_states * sizeof(double);
    hipMalloc(&d_eigenvalues, eigenvaluesSize);
    
    int* d_info = nullptr;
    hipMalloc(&d_info, num_states * sizeof(int));
    
    // 4. Query workspace size for batched eigen decomposition.
    int lwork = 0;
    hipsolverDnDsyevjBatched_bufferSize(_cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_LOWER,
                                       dim_conf, d_covariance, dim_conf, d_eigenvalues, &lwork, _syevj_params, num_states);
    
    // 5. Allocate workspace memory.
    double* work = nullptr;
    hipMalloc(&work, lwork * sizeof(double));
    
    // 6. Perform batched symmetric eigen decomposition.
    hipsolverStatus_t status = hipsolverDnDsyevjBatched(_cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_LOWER,
                           dim_conf, d_covariance, dim_conf, d_eigenvalues, work, lwork, d_info, _syevj_params, num_states);


    if (status != HIPSOLVER_STATUS_SUCCESS) {
        std::cerr << "hipsolverDnDsyevjBatched failed with status " << status << std::endl;
    }
    std::cout << "Time to perform eigen decomposition: " << timer.end_mus_output() << " us" << std::endl;
    
    // 7. Copy eigenvector matrices from d_covariance to d_eigvec (Preserve original eigenvectors)
    double* d_eigvec = nullptr;
    hipMalloc(&d_eigvec, covarianceSize);
    hipMemcpy(d_eigvec, d_covariance, covarianceSize, hipMemcpyDeviceToDevice);
    
    // 8. Launch combined kernel to compute sqrt of eigenvalues and scale eigenvectors.
    timer.start();
    double* d_scaledEigvec = nullptr;
    hipMalloc(&d_scaledEigvec, covarianceSize);
    int threadsPerBlock = 256;
    int blocksPerGrid = num_states;
    size_t sharedMemSize = dim_conf * sizeof(double);
    sqrtAndScaleEigenvectorsKernel<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_eigenvalues, d_eigvec, d_scaledEigvec, dim_conf, num_states);

    hipDeviceSynchronize();
    std::cout << "Time to compute sqrt and scale eigenvectors: " << timer.end_mus_output() << " us" << std::endl;
    
    // 9. Use hipblasDgemmBatched to compute sqrtP = d_scaledEigvec * (d_eigvec)^T for each state.
    timer.start();
    double* d_sqrtP = nullptr;
    size_t sqrtPSize = num_states * dim_conf * dim_conf * sizeof(double);
    hipMalloc(&d_sqrtP, sqrtPSize);
    
    std::vector<const double*> h_A_gemm(num_states);
    std::vector<const double*> h_B_gemm(num_states);
    std::vector<double*> h_C_gemm(num_states);
    for (int i = 0; i < num_states; i++) {
        h_A_gemm[i] = d_scaledEigvec + i * dim_conf * dim_conf; // scaled eigenvectors
        h_B_gemm[i] = d_eigvec + i * dim_conf * dim_conf;         // original eigenvectors
        h_C_gemm[i] = d_sqrtP + i * dim_conf * dim_conf;           // output sqrtP
    }
    const double** d_A_gemm = nullptr;
    const double** d_B_gemm = nullptr;
    double** d_C_gemm = nullptr;
    hipMalloc(&d_A_gemm, num_states * sizeof(const double*));
    hipMalloc(&d_B_gemm, num_states * sizeof(const double*));
    hipMalloc(&d_C_gemm, num_states * sizeof(double*));
    hipMemcpy(d_A_gemm, h_A_gemm.data(), num_states * sizeof(const double*), hipMemcpyHostToDevice);
    hipMemcpy(d_B_gemm, h_B_gemm.data(), num_states * sizeof(const double*), hipMemcpyHostToDevice);
    hipMemcpy(d_C_gemm, h_C_gemm.data(), num_states * sizeof(double*), hipMemcpyHostToDevice);    
    
    // Compute sqrtP = d_scaledEigvec * (d_eigvec)^T for each state.
    hipblasDgemmBatched(_cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T,
                       dim_conf, dim_conf, dim_conf,
                       &alpha,
                       d_A_gemm, dim_conf,
                       d_B_gemm, dim_conf,
                       &beta,
                       d_C_gemm, dim_conf,
                       num_states);
    std::cout << "Time to compute sqrtP: " << timer.end_mus_output() << " us" << std::endl;
    
    // 10. Compute sigmapts for each state: sigmapts = _zeromean_gpu * (sqrtP)^T.
    timer.start();
    std::vector<const double*> h_A2_gemm(num_states);
    std::vector<const double*> h_B2_gemm(num_states);
    std::vector<double*> h_C2_gemm(num_states);
    for (int i = 0; i < num_states; i++) {
        h_A2_gemm[i] = _zeromean_gpu; 
        h_B2_gemm[i] = d_sqrtP + i * dim_conf * dim_conf; // sqrtP for current state
        h_C2_gemm[i] = d_sigmapts + i * _sigmapts_rows * dim_conf;
    }
    const double** d_A2_gemm = nullptr;
    const double** d_B2_gemm = nullptr;
    double** d_C2_gemm = nullptr;
    hipMalloc(&d_A2_gemm, num_states * sizeof(const double*));
    hipMalloc(&d_B2_gemm, num_states * sizeof(const double*));
    hipMalloc(&d_C2_gemm, num_states * sizeof(double*));
    hipMemcpy(d_A2_gemm, h_A2_gemm.data(), num_states * sizeof(const double*), hipMemcpyHostToDevice);
    hipMemcpy(d_B2_gemm, h_B2_gemm.data(), num_states * sizeof(const double*), hipMemcpyHostToDevice);
    hipMemcpy(d_C2_gemm, h_C2_gemm.data(), num_states * sizeof(double*), hipMemcpyHostToDevice);
    
    // Compute sigmapts = _zeromean_gpu * (sqrtP)^T.
    hipblasDgemmBatched(_cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T,
                       _sigmapts_rows, dim_conf, dim_conf,
                       &alpha,
                       d_A2_gemm, _sigmapts_rows,
                       d_B2_gemm, dim_conf,
                       &beta,
                       d_C2_gemm, _sigmapts_rows,
                       num_states);
    std::cout << "Time to compute sigmapts: " << timer.end_mus_output() << " us" << std::endl;
    
    // 11. Add the mean to each state's sigmapts block. (Several milliseconds)
    int totalElements = num_states * _sigmapts_rows * dim_conf;
    int threads = 256;
    int blocks = (totalElements + threads - 1) / threads;
    addMeanKernel_batched<<<blocks, threads>>>(d_sigmapts, d_mean, _sigmapts_rows, dim_conf, num_states);
    
    // // 12. Copy the final sigmapts from device to host.
    // timer.start();
    // Eigen::MatrixXd sigma(_sigmapts_rows, dim_conf * num_states);
    // hipMemcpy(sigma.data(), d_sigmapts, sigmaptsSize, hipMemcpyDeviceToHost);
    // // sigmapts.resize(_sigmapts_rows, dim_conf * num_states);
    // sigmapts = sigma;
    // std::cout << "Time to copy sigmapts to host: " << timer.end_mus_output() << " us" << std::endl;
    
    // 13. Free all allocated device memory and destroy handles.
    timer.start();
    hipFree(d_covariance);
    hipFree(d_mean);
    hipFree(d_sigmapts);
    hipFree(d_eigenvalues);
    hipFree(d_info);
    hipFree(work);
    hipFree(d_eigvec);
    hipFree(d_scaledEigvec);
    hipFree(d_sqrtP);
    
    hipFree(d_A_gemm);
    hipFree(d_B_gemm);
    hipFree(d_C_gemm);
    hipFree(d_A2_gemm);
    hipFree(d_B2_gemm);
    hipFree(d_C2_gemm);
    
    // hipsolverDnDestroySyevjInfo(_syevj_params);
    // hipsolverDnDestroy(_cusolverH);
    // // hipblasDestroy(_cublasH);
    std::cout << "Time to free memory and destroy handles: " << timer.end_mus_output() << " us" << std::endl;
}



template <typename SDFType>
void CudaOperation_Base<SDFType>::initializeSigmaptsResources(int dim_conf, int num_states, int sigmapts_rows){
    printGPUMemoryInfo();
    // Allocate global device memory for covariance, mean, and sigmapts.
    std::cout << "sigmaPts rows: " << sigmapts_rows << " , sigmarows: " << _sigmapts_rows << std::endl;
    hipMalloc(&covariance_gpu, dim_conf * dim_conf * num_states * sizeof(double));
    hipMalloc(&d_sigmapt_cuda, sigmapts_rows * dim_conf * num_states * sizeof(double));
    hipMalloc(&mean_gpu, 2 * dim_conf * (num_states+2) * sizeof(double));

    num_streams = 5;
    
    // Resize vectors to hold per-state resources.
    streams.resize(num_streams);
    cusolver_handles.resize(num_streams);
    cublas_handles.resize(num_streams);

    for (int i = 0; i < num_streams; i++) {
        // Create CUDA stream.
        hipStreamCreate(&streams[i]);
        // Create cuSOLVER and cuBLAS handles and bind them with the stream.
        hipsolverDnCreate(&cusolver_handles[i]);
        hipblasCreate(&cublas_handles[i]);
        hipsolverSetStream(cusolver_handles[i], streams[i]);
        hipblasSetStream(cublas_handles[i], streams[i]);
    }

    // // Try one thread first
    // hipsolverDnCreate(&cusolver_handle);
    // hipblasCreate(&cublas_handle);

    d_eigen_values_vec.resize(num_states, nullptr);
    d_info_vec.resize(num_states, nullptr);
    d_work_vec.resize(num_states, nullptr);
    Lwork_vec.resize(num_states, 0);
    d_V_scaled_vec.resize(num_states, nullptr);
    d_sqrtP_vec.resize(num_states, nullptr);

    double* d_dummy;
    hipMalloc(&d_dummy, dim_conf * dim_conf * sizeof(double));
    
    // For each state, create stream, create handles, and allocate temporary memory.
    for (int state = 0; state < num_states; state++) {
        int stream_idx = state % num_streams;
        // Allocate memory for eigenvalues and info.
        hipMalloc(&d_eigen_values_vec[state], dim_conf * sizeof(double));
        hipMalloc(&d_info_vec[state], sizeof(int));
        
        // Query workspace size for eigen decomposition.
        hipsolverDnDsyevd_bufferSize(cusolver_handles[stream_idx],
                                    HIPSOLVER_EIG_MODE_VECTOR,
                                    HIPBLAS_FILL_MODE_LOWER,
                                    dim_conf,
                                    d_dummy,
                                    dim_conf,
                                    d_eigen_values_vec[state],
                                    &Lwork_vec[state]);
        // Allocate workspace memory.
        hipMalloc(&d_work_vec[state], Lwork_vec[state] * sizeof(double));
        
        // Allocate memory for V_scaled and sqrtP matrices (each size: dim_conf x dim_conf).
        hipMalloc(&d_V_scaled_vec[state], dim_conf * dim_conf * sizeof(double));
        hipMalloc(&d_sqrtP_vec[state], dim_conf * dim_conf * sizeof(double));
    }
    hipFree(d_dummy);

    printGPUMemoryInfo();
}


template <typename SDFType>
void CudaOperation_Base<SDFType>::update_sigmapts_separate(const MatrixXd& covariance, const MatrixXd& mean, int dim_conf, int num_states, MatrixXd& sigmapts){
    // printGPUMemoryInfo();
    const double alpha = 1.0, beta = 0.0;

    Timer timer;
    
    // Copy new covariance and mean data into the pre-allocated device memory.
    size_t covariance_size = dim_conf * dim_conf * num_states * sizeof(double);
    size_t mean_size = mean.size() * sizeof(double);
    hipMemcpy(covariance_gpu, covariance.data(), covariance_size, hipMemcpyHostToDevice);
    hipMemcpy(mean_gpu, mean.data(), mean_size, hipMemcpyHostToDevice);
    
    // For each state, perform the computation.
    for (int state = 0; state < num_states; state++) {
        // Each state's covariance matrix is stored at an offset in covariance_gpu.
        double* d_Pi = covariance_gpu + state * dim_conf * dim_conf;
        // Assuming mean data occupies two rows per state; adjust the offset as needed.
        double* d_mi = mean_gpu + 2 * (state + 1) * dim_conf;

        int stream_idx = state % num_streams;
        
        if (state % 1000 == 0){
            timer.start();
        }
        
        // 2.1 Perform symmetric eigenvalue decomposition on d_Pi.
        hipsolverDnDsyevd(cusolver_handles[stream_idx],
                         HIPSOLVER_EIG_MODE_VECTOR,
                         HIPBLAS_FILL_MODE_LOWER,
                         dim_conf,
                         d_Pi,
                         dim_conf,
                         d_eigen_values_vec[state],
                         d_work_vec[state],
                         Lwork_vec[state],
                         d_info_vec[state]);

        if (state % 1000 == 0){
            std::cout << "Eigen decomposition time: " << timer.end_mus_output() << " us" << std::endl;
            timer.start();
        }
        
        int h_info;
        hipMemcpyAsync(&h_info, d_info_vec[state], sizeof(int), hipMemcpyDeviceToHost);
        if (h_info != 0) {
            std::cerr << "Eigen decomposition failed for state " << state << " with info " << h_info << std::endl;
        }

        // 2.2 Apply square root to the eigenvalues using a custom kernel.
        int threadsPerBlock = 8;
        int blocks = (dim_conf + threadsPerBlock - 1) / threadsPerBlock;
        sqrtKernel<<<blocks, threadsPerBlock, 0, streams[stream_idx]>>>(d_eigen_values_vec[state], dim_conf);
        hipStreamSynchronize(streams[stream_idx]);

        if (state % 1000 == 0){
            std::cout << "Sqrt kernel time: " << timer.end_mus_output() << " us" << std::endl;
            timer.start();
        }
        
        // 2.3 Use hipblasDdgmm to scale the eigenvector matrix by the square-rooted eigenvalues.
        hipblasStatus_t stat = hipblasDdgmm(cublas_handles[stream_idx], HIPBLAS_SIDE_RIGHT, dim_conf, dim_conf,
                                          d_Pi, dim_conf,
                                          d_eigen_values_vec[state], 1,
                                          d_V_scaled_vec[state], dim_conf);
        if (stat != HIPBLAS_STATUS_SUCCESS) {
            std::cerr << "hipblasDdgmm failed for state " << state << ": " << stat << std::endl;
        }

        if (state % 1000 == 0){
            std::cout << "Ddgmm time: " << timer.end_mus_output() << " us" << std::endl;
            timer.start();
        }
        
        // 2.4 Compute sqrtP = d_V_scaled * (d_Pi)^T.
        stat = hipblasDgemm(cublas_handles[stream_idx], HIPBLAS_OP_N, HIPBLAS_OP_T, 
                           dim_conf, dim_conf, dim_conf,
                           &alpha, d_V_scaled_vec[state], dim_conf,
                           d_Pi, dim_conf,
                           &beta, d_sqrtP_vec[state], dim_conf);
        if (stat != HIPBLAS_STATUS_SUCCESS) {
            std::cerr << "hipblasDgemm for sqrtP failed for state " << state << ": " << stat << std::endl;
        }

        if (state % 1000 == 0){
            std::cout << "Gemm time: " << timer.end_mus_output() << " us" << std::endl;
            timer.start();
        }
        
        // 2.5 Compute the current state's portion of sigmapts.
        // d_sigmapt_cuda is allocated contiguously on the device; each state occupies a block of _sigmapts_rows x dim_conf.
        
        // std::cout << "sigma rows: " << _sigmapts_rows << " dim_conf: " << dim_conf << " state: " << state << std::endl;
        double* d_sigmapts_state = d_sigmapt_cuda + state * _sigmapts_rows * dim_conf;
        // hipMemcpy(_zeromean_gpu, zeromean.data(), _sigmapts_rows * dim_conf * sizeof(double), hipMemcpyHostToDevice);
        stat = hipblasDgemm(cublas_handles[stream_idx], HIPBLAS_OP_N, HIPBLAS_OP_T, 
                           _sigmapts_rows, dim_conf, dim_conf,
                           &alpha, _zeromean_gpu, _sigmapts_rows,
                           d_sqrtP_vec[state], dim_conf,
                           &beta, d_sigmapts_state, _sigmapts_rows);
        if (stat != HIPBLAS_STATUS_SUCCESS) {
            std::cerr << "hipblasDgemm for sigmapts failed for state " << state << ": " << stat << std::endl;
        }

        if (state % 1000 == 0){
            std::cout << "Zero mean time: " << timer.end_mus_output() << " us" << std::endl;
            timer.start();
        }
        
        // 2.6 Add the mean to sigmapts (for the current state's block).
        threadsPerBlock = 256;
        blocks = (_sigmapts_rows * dim_conf + threadsPerBlock - 1) / threadsPerBlock;
        addMeanKernel<<<blocks, threadsPerBlock, 0, streams[stream_idx]>>>(d_sigmapts_state, d_mi, _sigmapts_rows, dim_conf);

        if (state % 1000 == 0){
            std::cout << "Add mean time: " << timer.end_mus_output() << " us" << std::endl << std::endl;
        }
    }
    
    // Synchronize all streams.
    for (int i = 0; i < num_streams; i++) {
        hipStreamSynchronize(streams[i]);
    }
    
    // // Copy the computed sigmapts from device to host.
    // MatrixXd sigma(_sigmapts_rows, dim_conf * num_states);
    // size_t sigmapts_size = _sigmapts_rows * dim_conf * num_states * sizeof(double);
    // hipMemcpy(sigma.data(), d_sigmapt_cuda, sigmapts_size, hipMemcpyDeviceToHost);
    // sigmapts = sigma;
}

template <typename SDFType>
void CudaOperation_Base<SDFType>::freeSigmaptsResources(int num_states)
{
    // std::cout << "Memory before freeing: ";
    // printGPUMemoryInfo();

    // Free per-state temporary memory and destroy handles/streams.
    for (int state = 0; state < num_states; state++) {
        hipFree(d_eigen_values_vec[state]);
        hipFree(d_info_vec[state]);
        hipFree(d_work_vec[state]);
        hipFree(d_V_scaled_vec[state]);
        hipFree(d_sqrtP_vec[state]);
    }
    // Free global device memory.
    hipFree(covariance_gpu);
    hipFree(mean_gpu);
    hipFree(d_sigmapt_cuda);

    for(int i = 0; i < num_streams; i++){
        hipsolverDnDestroy(cusolver_handles[i]);
        // hipblasDestroy(cublas_handles[i]);
        hipStreamDestroy(streams[i]);
    }
    
    // Clear the vectors.
    streams.clear();
    cusolver_handles.clear();
    cublas_handles.clear();
    d_eigen_values_vec.clear();
    d_info_vec.clear();
    d_work_vec.clear();
    Lwork_vec.clear();
    d_V_scaled_vec.clear();
    d_sqrtP_vec.clear();

    // std::cout << "Memory after freeing: ";
    // printGPUMemoryInfo();
    
}



void CudaOperation_PlanarPR::CudaIntegration(const MatrixXd& sigmapts, const MatrixXd& weights, MatrixXd& results, const MatrixXd& mean, int type)
{
    double *sigmapts_gpu, *mu_gpu, *pts_gpu, *result_gpu;
    int n_balls = 1;

    hipMalloc(&sigmapts_gpu, sigmapts.size() * sizeof(double));
    hipMalloc(&mu_gpu, sigmapts.cols() * sizeof(double));
    hipMalloc(&pts_gpu, sigmapts.rows() * results.size() * sizeof(double));
    hipMalloc(&result_gpu, results.size() * sizeof(double));

    hipMemcpy(sigmapts_gpu, sigmapts.data(), sigmapts.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(mu_gpu, mean.data(), sigmapts.cols() * sizeof(double), hipMemcpyHostToDevice);

    // Kernel 1: Obtain the result of function 
    dim3 blockSize1(256, 256);
    dim3 threadperblock1((results.cols()*sigmapts.rows() + blockSize1.x - 1) / blockSize1.x, (results.rows() + blockSize1.y - 1) / blockSize1.y);

    Sigma_function<<<blockSize1, threadperblock1>>>(sigmapts_gpu, pts_gpu, mu_gpu, sigmapts.rows(), sigmapts.cols(), results.rows(), results.cols(), type, _class_gpu, _data_gpu);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(sigmapts_gpu);
    hipFree(mu_gpu);

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 blockSize2(256, 256);
    dim3 threadperblock2((results.cols() + blockSize2.x - 1) / blockSize2.x, (results.rows() + blockSize2.y - 1) / blockSize2.y);

    obtain_res<<<blockSize2, threadperblock2>>>(pts_gpu, _weight_gpu, result_gpu, sigmapts.rows(), results.rows(), results.cols());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(pts_gpu);
    hipFree(result_gpu);
}

void CudaOperation_PlanarPR::costIntegration(const MatrixXd& sigmapts, VectorXd& results, const int sigmapts_cols){
    double *result_gpu;
    hipMalloc(&result_gpu, results.size() * sizeof(double));

    // Kernel 1: Obtain the result of function 
    dim3 threadperblock1(32, 32);
    dim3 blockSize1((results.size() + threadperblock1.x - 1) / threadperblock1.x, (sigmapts.rows() + threadperblock1.y - 1) / threadperblock1.y);

    cost_function<<<blockSize1, threadperblock1>>>(_sigmapts_gpu, _func_value_gpu, sigmapts.rows(), sigmapts_cols, results.size(), _class_gpu, _data_gpu);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 threadperblock2(256);
    dim3 blockSize2((results.size() + threadperblock2.x - 1) / threadperblock2.x);

    obtain_cost<<<blockSize2, threadperblock2>>>(_func_value_gpu, _weight_gpu, result_gpu, sigmapts.rows(), results.size());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(result_gpu);
}


void CudaOperation_3dpR::CudaIntegration(const MatrixXd& sigmapts, const MatrixXd& weights, MatrixXd& results, const MatrixXd& mean, int type)
{
    double *sigmapts_gpu, *mu_gpu, *pts_gpu, *result_gpu;
    int n_balls = 1;

    hipMalloc(&sigmapts_gpu, sigmapts.size() * sizeof(double));
    hipMalloc(&mu_gpu, sigmapts.cols() * sizeof(double));
    hipMalloc(&pts_gpu, sigmapts.rows() * results.size() * sizeof(double));
    hipMalloc(&result_gpu, results.size() * sizeof(double));

    hipMemcpy(sigmapts_gpu, sigmapts.data(), sigmapts.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(mu_gpu, mean.data(), sigmapts.cols() * sizeof(double), hipMemcpyHostToDevice);

    // Kernel 1: Obtain the result of function 
    dim3 blockSize1(64, 64);
    dim3 threadperblock1((results.cols()*sigmapts.rows() + blockSize1.x - 1) / blockSize1.x, (results.rows() + blockSize1.y - 1) / blockSize1.y);

    Sigma_function<<<blockSize1, threadperblock1>>>(sigmapts_gpu, pts_gpu, mu_gpu, sigmapts.rows(), sigmapts.cols(), results.rows(), results.cols(), type, _class_gpu, _data_gpu);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(sigmapts_gpu);
    hipFree(mu_gpu);
    

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 blockSize2(64, 64);
    dim3 threadperblock2((results.cols() + blockSize2.x - 1) / blockSize2.x, (results.rows() + blockSize2.y - 1) / blockSize2.y);

    obtain_res<<<blockSize2, threadperblock2>>>(pts_gpu, _weight_gpu, result_gpu, sigmapts.rows(), results.rows(), results.cols());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(pts_gpu);
    hipFree(result_gpu);
}

void CudaOperation_3dpR::costIntegration(const MatrixXd& sigmapts, VectorXd& results, const int sigmapts_cols){
    double *result_gpu;

    hipMalloc(&result_gpu, results.size() * sizeof(double));

    // Kernel 1: Obtain the result of function 
    dim3 blockSize1(64, 64);
    dim3 threadperblock1((results.size() + blockSize1.x - 1) / blockSize1.x, (sigmapts.rows() + blockSize1.y - 1) / blockSize1.y);

    cost_function<<<blockSize1, threadperblock1>>>(_sigmapts_gpu, _func_value_gpu, sigmapts.rows(), sigmapts_cols, results.size(), _class_gpu, _data_gpu);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 blockSize2(64);
    dim3 threadperblock2((results.size() + blockSize2.x - 1) / blockSize2.x);

    obtain_cost<<<blockSize2, threadperblock2>>>(_func_value_gpu, _weight_gpu, result_gpu, sigmapts.rows(), results.size());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(result_gpu);
}


void CudaOperation_Quad::CudaIntegration(const MatrixXd& sigmapts, const MatrixXd& weights, MatrixXd& results, const MatrixXd& mean, int type)
{
    double *sigmapts_gpu, *pts_gpu, *result_gpu, *mu_gpu;
    int n_balls = 5;

    hipMalloc(&sigmapts_gpu, sigmapts.size() * sizeof(double));
    hipMalloc(&pts_gpu, sigmapts.rows() * results.size() * sizeof(double));
    hipMalloc(&result_gpu, results.size() * sizeof(double));
    hipMalloc(&mu_gpu, sigmapts.cols() * sizeof(double));

    hipMemcpy(sigmapts_gpu, sigmapts.data(), sigmapts.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(mu_gpu, mean.data(), sigmapts.cols() * sizeof(double), hipMemcpyHostToDevice);

    // Kernel 1: Obtain the result of function 
    dim3 blockSize1(64, 64);
    dim3 threadperblock1((results.cols()*sigmapts.rows() + blockSize1.x - 1) / blockSize1.x, (results.rows() + blockSize1.y - 1) / blockSize1.y);

    Sigma_function<<<blockSize1, threadperblock1>>>(sigmapts_gpu, pts_gpu, mu_gpu, sigmapts.rows(), sigmapts.cols(), results.rows(), results.cols(), type, _class_gpu, _data_gpu);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(sigmapts_gpu);
    hipFree(mu_gpu);

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 blockSize2(64, 64);
    dim3 threadperblock2((results.cols() + blockSize2.x - 1) / blockSize2.x, (results.rows() + blockSize2.y - 1) / blockSize2.y);

    obtain_res<<<blockSize2, threadperblock2>>>(pts_gpu, _weight_gpu, result_gpu, sigmapts.rows(), results.rows(), results.cols());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(pts_gpu);
    hipFree(result_gpu);
}

void CudaOperation_Quad::costIntegration(const MatrixXd& sigmapts, VectorXd& results, const int sigmapts_cols){
    double *result_gpu;

    hipMalloc(&result_gpu, results.size() * sizeof(double));
    // hipMemcpy(_sigmapts_gpu, sigmapts.data(), sigmapts.size() * sizeof(double), hipMemcpyHostToDevice);

    // Kernel 1: Obtain the result of function 
    dim3 threadperblock1(32, 32);
    dim3 blockSize1((results.size() + threadperblock1.x - 1) / threadperblock1.x, (sigmapts.rows() + threadperblock1.y - 1) / threadperblock1.y);

    cost_function<<<blockSize1, threadperblock1>>>(_sigmapts_gpu, _func_value_gpu, sigmapts.rows(), sigmapts_cols, results.size(), _class_gpu, _data_gpu);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 threadperblock2(256);
    dim3 blockSize2((results.size() + threadperblock2.x - 1) / threadperblock2.x);

    obtain_cost<<<blockSize2, threadperblock2>>>(_func_value_gpu, _weight_gpu, result_gpu, sigmapts.rows(), results.size());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    // hipFree(_func_value_gpu);
    hipFree(result_gpu);
}

template <typename SDFType>
void CudaOperation_Base<SDFType>::dmuIntegration(const MatrixXd& sigmapts, const MatrixXd& mu, VectorXd& results, const int sigmapts_cols){

    double *vec_gpu, *result_gpu;
    hipMalloc(&vec_gpu, sigmapts.rows() * results.size() * sizeof(double));
    hipMalloc(&result_gpu, results.size() * sizeof(double));
    hipMalloc(&_mu_gpu, mu.size() * sizeof(double));

    hipMemcpy(_mu_gpu, mu.data(), mu.size() * sizeof(double), hipMemcpyHostToDevice);

    // Kernel 1: Obtain the result of function 
    dim3 threadperblock1(32, 32);
    dim3 blockSize1((results.size() + threadperblock1.x - 1) / threadperblock1.x, (_sigmapts_rows + threadperblock1.y - 1) / threadperblock1.y);

    dmu_function<<<blockSize1, threadperblock1>>>(_sigmapts_gpu, _mu_gpu, _func_value_gpu, vec_gpu, _sigmapts_rows, _dim_conf, _n_states);
    hipDeviceSynchronize();

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 threadperblock2(256);
    dim3 blockSize2((results.size() + threadperblock2.x - 1) / threadperblock2.x);

    obtain_dmu<<<blockSize2, threadperblock2>>>(vec_gpu, _weight_gpu, result_gpu, sigmapts.rows(), results.size());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(vec_gpu);
    hipFree(result_gpu);
}

template <typename SDFType>
void CudaOperation_Base<SDFType>::ddmuIntegration(MatrixXd& results){
    // Reuse the sigmapts and pts passed into gpu before
    double *vec_gpu, *result_gpu;
    hipMalloc(&vec_gpu, _sigmapts_rows * results.size() * sizeof(double));
    hipMalloc(&result_gpu, results.size() * sizeof(double));

    // Kernel 1: Obtain the result of function 
    dim3 threadperblock(32, 32); //1024
    dim3 blockSize1((results.cols() + threadperblock.x - 1) / threadperblock.x, (_sigmapts_rows * results.rows() + threadperblock.y - 1) / threadperblock.y);

    ddmu_function<<<blockSize1, threadperblock>>>(_sigmapts_gpu, _mu_gpu, _func_value_gpu, vec_gpu, _sigmapts_rows, _dim_conf, _n_states);
    hipDeviceSynchronize();

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 blockSize2((results.cols() + threadperblock.x - 1) / threadperblock.x, (results.rows() + threadperblock.y - 1) / threadperblock.y);

    obtain_ddmu<<<blockSize2, threadperblock>>>(vec_gpu, _weight_gpu, result_gpu, _sigmapts_rows, results.rows(), results.cols());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(vec_gpu);
    hipFree(result_gpu);
    hipFree(_mu_gpu);
}

void CudaOperation_3dArm::CudaIntegration(const MatrixXd& sigmapts, const MatrixXd& weights, MatrixXd& results, const MatrixXd& mean, int type)
{
    double *sigmapts_gpu, *mu_gpu, *pts_gpu, *result_gpu;
    int n_balls = 1;
    hipMalloc(&sigmapts_gpu, sigmapts.size() * sizeof(double));
    hipMalloc(&mu_gpu, sigmapts.cols() * sizeof(double));
    hipMalloc(&pts_gpu, sigmapts.rows() * results.size() * sizeof(double));
    hipMalloc(&result_gpu, results.size() * sizeof(double));

    hipMemcpy(sigmapts_gpu, sigmapts.data(), sigmapts.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(mu_gpu, mean.data(), sigmapts.cols() * sizeof(double), hipMemcpyHostToDevice);

    // Kernel 1: Obtain the result of function 
    dim3 blockSize1(1024, 1024);
    dim3 threadperblock1((results.cols()*sigmapts.rows() + blockSize1.x - 1) / blockSize1.x, (results.rows() + blockSize1.y - 1) / blockSize1.y);

    Sigma_function<<<blockSize1, threadperblock1>>>(sigmapts_gpu, pts_gpu, mu_gpu, sigmapts.rows(), sigmapts.cols(), results.rows(), results.cols(), type, _class_gpu, _data_gpu);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(sigmapts_gpu);
    hipFree(mu_gpu);
    

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 blockSize2(1024, 1024);
    dim3 threadperblock2((results.cols() + blockSize2.x - 1) / blockSize2.x, (results.rows() + blockSize2.y - 1) / blockSize2.y);

    obtain_res<<<blockSize2, threadperblock2>>>(pts_gpu, _weight_gpu, result_gpu, sigmapts.rows(), results.rows(), results.cols());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(pts_gpu);
    hipFree(result_gpu);
}

void CudaOperation_3dArm::costIntegration(const MatrixXd& sigmapts, VectorXd& results, const int sigmapts_cols){
    double *result_gpu;

    hipMalloc(&result_gpu, results.size() * sizeof(double));
    hipMemcpy(_sigmapts_gpu, sigmapts.data(), sigmapts.size() * sizeof(double), hipMemcpyHostToDevice);

    // Kernel 1: Obtain the result of function 
    dim3 threadperblock1(32, 32);
    dim3 blockSize1((results.size() + threadperblock1.x - 1) / threadperblock1.x, (sigmapts.rows() + threadperblock1.y - 1) / threadperblock1.y);

    cost_function<<<blockSize1, threadperblock1>>>(_sigmapts_gpu, _func_value_gpu, sigmapts.rows(), sigmapts_cols, results.size(), 
                                                    _class_gpu, _data_gpu, _a_gpu, _alpha_gpu, _d_gpu, _theta_gpu,
                                                    _rad_gpu, _frames_gpu, _centers_gpu);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 threadperblock2(256);
    dim3 blockSize2((results.size() + threadperblock2.x - 1) / threadperblock2.x);

    obtain_cost<<<blockSize2, threadperblock2>>>(_func_value_gpu, _weight_gpu, result_gpu, sigmapts.rows(), results.size());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(result_gpu);
}



// set m, l, J as input
__host__ __device__ void function_value(const VectorXd& sigmapt, VectorXd& function_value){
    double px = sigmapt(0);       // p_x
    double pz = sigmapt(1);       // p_z
    double phi = sigmapt(2);      // ϕ
    double vx = sigmapt(3);       // v_x
    double vz = sigmapt(4);       // v_z
    double phi_dot = sigmapt(5);  // ϕ_dot
    const double g = 9.81;

    function_value(0) = vx * cos(phi) - vz * sin(phi); // \(\dot{p_x}\)
    function_value(1) = vx * sin(phi) + vz * cos(phi); // \(\dot{p_z}\)
    function_value(2) = phi_dot;                       // \(\dot{\phi}\)
    function_value(3) = vz * phi_dot - g * sin(phi);   // \(\dot{v_x}\)
    function_value(4) = -vx * phi_dot - g * cos(phi);  // \(\dot{v_z}\)
    function_value(5) = 0.0;                           // \(\ddot{\phi}\)
}

__global__ void obtain_y_sigma(double* d_sigmapts, double* d_y_sigmapts, int sigmapts_rows, int sigmapts_cols, int n_states){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < sigmapts_rows && col < n_states){
        Eigen::Map<MatrixXd> sigmapts(d_sigmapts + col*sigmapts_rows*sigmapts_cols, sigmapts_rows, sigmapts_cols);
        VectorXd y_sigmapt(sigmapts_cols);
        function_value(sigmapts.row(row), y_sigmapt);
        for (int i = 0; i < sigmapts_cols; i++)
            d_y_sigmapts[col*sigmapts_rows*sigmapts_cols + i * sigmapts_rows + row] = y_sigmapt(i);
    }
}

__global__ void covariance_function(double* d_sigmapts, double* d_x_bar, double* d_y_sigmapts, double* d_y_bar, double* d_vec, int sigmapts_rows, int sigmapts_cols, int n_states){
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < sigmapts_rows*sigmapts_cols && col < n_states*sigmapts_cols){
        int idx_x = col / sigmapts_cols; // Use which x_bar and y_bar
        int idx_y = row / sigmapts_cols; // use which sigma point
        int mat_x = col % sigmapts_cols; // which element in x
        int mat_y = row % sigmapts_cols;

        d_vec[col*sigmapts_rows*sigmapts_cols + row] = (d_sigmapts[(idx_x*sigmapts_cols + mat_y) * sigmapts_rows + idx_y] - d_x_bar[idx_x*sigmapts_cols + mat_y]) 
                                                     * (d_y_sigmapts[(idx_x*sigmapts_cols + mat_x) * sigmapts_rows + idx_y] - d_y_bar[idx_x*sigmapts_cols + mat_x]) ;
    }
}

__global__ void obtain_covariance(double* d_vec, double* d_weights, double* d_result, int sigmapts_rows, int res_rows, int res_cols){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < res_rows && col < res_cols){
        double sum = 0;
        for(int i = 0; i < sigmapts_rows; i++){
            sum += d_vec[col*sigmapts_rows*res_rows + row + i*res_rows] * d_weights[i];
        }
        d_result[col*res_rows + row] = sum;
    }
}

__global__ void obtain_y_bar(double* d_pts, double* d_weights, double* d_result, int sigmapts_rows, int n_states){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n_states){
        double sum = 0;
        for(int i = 0; i < sigmapts_rows; i++){
            sum += d_pts[idx*sigmapts_rows + i]* d_weights[i];
        }
        d_result[idx] = sum;
    }
}

void CudaOperation_SLR::expectationIntegration(MatrixXd& y_bar){

    // Kernel 1: Obtain the result of function 
    dim3 threadperblock1(32, 32);
    dim3 blockSize1((_n_states + threadperblock1.x - 1) / threadperblock1.x, (_sigmapts_rows + threadperblock1.y - 1) / threadperblock1.y);

    obtain_y_sigma<<<blockSize1, threadperblock1>>>(_sigmapts_gpu, _y_sigmapts_gpu, _sigmapts_rows, _dim_state, _n_states);
    hipDeviceSynchronize();

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 threadperblock2(256);
    dim3 blockSize2((_dim_state*_n_states + threadperblock2.x - 1) / threadperblock2.x);

    obtain_y_bar<<<blockSize2, threadperblock2>>>(_y_sigmapts_gpu, _weights_gpu, _y_bar_gpu, _sigmapts_rows, _dim_state*_n_states);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(y_bar.data(), _y_bar_gpu, y_bar.size() * sizeof(double), hipMemcpyDeviceToHost);
}

void CudaOperation_SLR::covarianceIntegration(MatrixXd& results){
    // result is the matrix of P_xy (dim_state, dim_state*n_states)
    double *vec_gpu, *result_gpu;
    hipMalloc(&vec_gpu, _sigmapts_rows * results.size() * sizeof(double));
    hipMalloc(&result_gpu, results.size() * sizeof(double));

    // Kernel 1: Obtain the result of function 
    dim3 threadperblock(32, 32);
    dim3 blockSize1((results.cols() + threadperblock.x - 1) / threadperblock.x, (_sigmapts_rows * results.rows() + threadperblock.y - 1) / threadperblock.y);

    covariance_function<<<blockSize1, threadperblock>>>(_sigmapts_gpu, _x_bar_gpu, _y_sigmapts_gpu, _y_bar_gpu, vec_gpu, _sigmapts_rows, _dim_state, _n_states);
    hipDeviceSynchronize();

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 blockSize2((results.cols() + threadperblock.x - 1) / threadperblock.x, (results.rows() + threadperblock.y - 1) / threadperblock.y);

    obtain_covariance<<<blockSize2, threadperblock>>>(vec_gpu, _weights_gpu, result_gpu, _sigmapts_rows, _dim_state, _dim_state*_n_states);
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(vec_gpu);
    hipFree(result_gpu);
}





__global__ void compute_AT_B_A_kernel(const double* d_Mat_A, const double* d_Mat_B, double* d_result, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < cols && col < cols) {
        double sum = 0.0;
        for (int k = 0; k < rows; k++) {
            for (int l = 0; l < rows; l++) {
                sum += d_Mat_A[k * cols + row] * d_Mat_B[k * rows + l] * d_Mat_A[l * cols + col];
            }
        }
        d_result[row * cols + col] = sum;
    }
}

MatrixXd compute_AT_B_A(MatrixXd& Matrix_A, MatrixXd& Matrix_B){
    int rows = Matrix_A.rows();
    int cols = Matrix_A.cols();

    double *d_Mat_A, *d_Mat_B, *d_result;
    hipMalloc(&d_Mat_A, Matrix_A.size() * sizeof(double));
    hipMalloc(&d_Mat_B, Matrix_B.size() * sizeof(double));
    hipMalloc(&d_result, cols * cols * sizeof(double));

    hipMemcpy(d_Mat_A, Matrix_A.data(), Matrix_A.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Mat_B, Matrix_B.data(), Matrix_B.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemset(d_result, 0, cols * cols * sizeof(double));

    dim3 blockSize(16, 16);
    dim3 threadperblock((cols + blockSize.x - 1) / blockSize.x, (cols + blockSize.y - 1) / blockSize.y);

    compute_AT_B_A_kernel<<<blockSize, threadperblock>>>(d_Mat_A, d_Mat_B, d_result, rows, cols);

    MatrixXd result = MatrixXd::Zero(cols, cols);
    hipMemcpy(result.data(), d_result, cols * cols * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_Mat_A);
    hipFree(d_Mat_B);
    hipFree(d_result);

    return result;
}

__global__ void computeTmpKernel(double* tmp, const double* covariance, const double* AT_precision_A, int dim) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < dim && j < dim) {
        double sum = 0.0;
        for (int k = 0; k < dim; k++) {
            for (int l = 0; l < dim; l++) {
                sum += (covariance[i * dim + j] * covariance[k * dim + l] +
                        covariance[i * dim + k] * covariance[j * dim + l] +
                        covariance[i * dim + l] * covariance[j * dim + k]) * AT_precision_A[k * dim + l];
            }
        }
        tmp[i * dim + j] = sum;
    }
}



void computeTmp_CUDA(Eigen::MatrixXd& tmp, const Eigen::MatrixXd& covariance, const Eigen::MatrixXd& AT_precision_A){
    int dim = covariance.rows();
    double *d_tmp, *d_covariance, *d_AT_precision_A;
    hipMalloc(&d_tmp, dim * dim * sizeof(double));
    hipMalloc(&d_covariance, dim * dim * sizeof(double));
    hipMalloc(&d_AT_precision_A, dim * dim * sizeof(double));

    hipMemcpy(d_covariance, covariance.data(), dim * dim * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_AT_precision_A, AT_precision_A.data(), dim * dim * sizeof(double), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((dim + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (dim + threadsPerBlock.y - 1) / threadsPerBlock.y);

    computeTmpKernel<<<blocksPerGrid, threadsPerBlock>>>(d_tmp, d_covariance, d_AT_precision_A, dim);
    hipDeviceSynchronize();

    hipMemcpy(tmp.data(), d_tmp, dim * dim * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_tmp);
    hipFree(d_covariance);
    hipFree(d_AT_precision_A);
}


}


// hipDeviceProp_t prop;
// hipGetDeviceProperties(&prop, 0);
// std::cout << "Double precision performance ratio: " << prop.singleToDoublePrecisionPerfRatio << std::endl;



// // Using MAGMA batched interface to compute the square root of each state's covariance matrix,
// // then multiply with a given matrix (_zeromean_gpu) and finally add a mean vector.
// // - Assume covariance is a (dim_conf x dim_conf*num_states) matrix,
// // - where each state's covariance is stored consecutively (column-major);
// // - mean is a (num_states x dim_conf) matrix.
// // - The output sigmapts has size (_sigmapts_rows x dim_conf*num_states).
// template <typename SDFType>
// void CudaOperation_Base<SDFType>::update_sigmapts_magma_batched(const MatrixXd& covariance,
//                                                                   const MatrixXd& mean,
//                                                                   int dim_conf,
//                                                                   int num_states,
//                                                                   MatrixXd& sigmapts)
// {
//     int sig_rows = _sigmapts_rows;  // Predefined number of sigma points rows

//     // 1. Copy covariance and mean to device memory.
//     double *d_cov, *d_mean, *d_sig;
//     size_t cov_size = num_states * dim_conf * dim_conf * sizeof(double);
//     size_t mean_size = num_states * dim_conf * sizeof(double);
//     size_t sig_size = num_states * sig_rows * dim_conf * sizeof(double);
//     hipMalloc(&d_cov, cov_size);
//     hipMalloc(&d_mean, mean_size);
//     hipMalloc(&d_sig, sig_size);
//     hipMemcpy(d_cov, covariance.data(), cov_size, hipMemcpyHostToDevice);
//     hipMemcpy(d_mean, mean.data(), mean_size, hipMemcpyHostToDevice);

//     // 2. Construct a batched pointer array that points to each state's covariance matrix.
//     double **d_A_array;
//     hipMalloc(&d_A_array, num_states * sizeof(double*));
//     std::vector<double*> h_A_array(num_states);
//     for (int state = 0; state < num_states; state++) {
//         h_A_array[state] = d_cov + state * dim_conf * dim_conf;
//     }
//     hipMemcpy(d_A_array, h_A_array.data(), num_states * sizeof(double*), hipMemcpyHostToDevice);

//     // 3. Allocate device memory for eigenvalues (each state has dim_conf eigenvalues)
//     double *d_w;
//     hipMalloc(&d_w, num_states * dim_conf * sizeof(double));
//     // Allocate info array for each state.
//     int *d_info;
//     hipMalloc(&d_info, num_states * sizeof(int));

//     // 4. Create a MAGMA queue (MAGMA uses an internal CUDA stream).
//     magma_queue_t queue;
//     magma_queue_create(0, &queue);

//     // 5. Call MAGMA batched eigenvalue decomposition:
//     // Compute eigenvalue decomposition A = Q and store eigenvalues in d_w.

//     // // There is no such function
//     // magma_dsyevd_batched(MagmaVec, MagmaLower, dim_conf,
//     //                      d_A_array, dim_conf,
//     //                      d_w, d_info, num_states, queue);

//     magma_queue_sync(queue);

//     // 6. Scale each eigenvector column: Q(:,j) = Q(:,j) * sqrt(eigenvalue_j).
//     {
//         dim3 threads(16, 16);
//         dim3 grid((dim_conf + threads.x - 1) / threads.x,
//                   (num_states + threads.y - 1) / threads.y);
//         scale_eigvecs_kernel<<<grid, threads, 0, magma_queue_get_cuda_stream(queue)>>>(
//             d_A_array, d_w, dim_conf, num_states);
//         magma_queue_sync(queue);
//     }

//     // 7. Use batched GEMM to compute the square root of each covariance matrix:
//     // sqrt(P) = Q * Q^T for each state.
//     double **d_sqrtP_array;
//     hipMalloc(&d_sqrtP_array, num_states * sizeof(double*));
//     std::vector<double*> h_sqrtP_array(num_states);
//     // Allocate memory for sqrtP for each state (size: dim_conf x dim_conf).
//     double *d_sqrtP;
//     hipMalloc(&d_sqrtP, num_states * dim_conf * dim_conf * sizeof(double));
//     for (int s = 0; s < num_states; s++) {
//         h_sqrtP_array[s] = d_sqrtP + s * dim_conf * dim_conf;
//     }
//     hipMemcpy(d_sqrtP_array, h_sqrtP_array.data(), num_states * sizeof(double*), hipMemcpyHostToDevice);

//     double alpha = 1.0, beta = 0.0;
//     magma_dgemm_batched(MagmaNoTrans, MagmaTrans,
//                         dim_conf, dim_conf, dim_conf,
//                         alpha, d_A_array, dim_conf,
//                         d_A_array, dim_conf,
//                         beta, d_sqrtP_array, dim_conf,
//                         num_states, queue);
//     magma_queue_sync(queue);

//     // 8. Compute sigma points for each state using sqrt(P):
//     // For each state, compute: sigmapts_state = (_zeromean) * sqrt(P)
//     // _zeromean_gpu is assumed to be pre-allocated on the device (size: sig_rows x dim_conf).
//     // Construct a batched pointer array pointing to _zeromean (assumed identical for all states).
//     double **d_zeromean_array;
//     hipMalloc(&d_zeromean_array, num_states * sizeof(double*));
//     std::vector<double*> h_zeromean_array(num_states, _zeromean_gpu);
//     hipMemcpy(d_zeromean_array, h_zeromean_array.data(), num_states * sizeof(double*), hipMemcpyHostToDevice);

//     // Construct a batched pointer array for each state's output sigmapts block.
//     double **d_sig_array;
//     hipMalloc(&d_sig_array, num_states * sizeof(double*));
//     std::vector<double*> h_sig_array(num_states);
//     for (int s = 0; s < num_states; s++) {
//         h_sig_array[s] = d_sig + s * (sig_rows * dim_conf);
//     }
//     hipMemcpy(d_sig_array, h_sig_array.data(), num_states * sizeof(double*), hipMemcpyHostToDevice);

//     // Perform batched GEMM: for each state,
//     // sigmapts = _zeromean_gpu (size: sig_rows x dim_conf) multiplied by sqrt(P) (dim_conf x dim_conf).
//     magma_dgemm_batched(MagmaNoTrans, MagmaNoTrans,
//                         sig_rows, dim_conf, dim_conf,
//                         alpha, d_zeromean_array, sig_rows,
//                         d_sqrtP_array, dim_conf,
//                         beta, d_sig_array, sig_rows,
//                         num_states, queue);
//     magma_queue_sync(queue);

//     // 9. For each state, add the mean vector to the corresponding sigmapts (add to every row).
//     {
//         dim3 threads(16, 16);
//         // Total rows: num_states * sig_rows, each row has dim_conf elements.
//         dim3 grid((dim_conf + threads.x - 1) / threads.x,
//                   ((num_states * sig_rows) + threads.y - 1) / threads.y);
//         add_mean_kernel_batched<<<grid, threads, 0, magma_queue_get_cuda_stream(queue)>>>(
//             d_sig, d_mean, sig_rows, dim_conf, num_states);
//         magma_queue_sync(queue);
//     }

//     // 10. Copy the computed sigma points back to host.
//     sigmapts.resize(sig_rows, num_states * dim_conf);
//     hipMemcpy(sigmapts.data(), d_sig, sig_size, hipMemcpyDeviceToHost);

//     // 11. Free all device resources.
//     hipFree(d_cov);
//     hipFree(d_mean);
//     hipFree(d_sig);
//     hipFree(d_A_array);
//     hipFree(d_w);
//     hipFree(d_info);
//     hipFree(d_sqrtP_array);
//     hipFree(d_sqrtP);
//     hipFree(d_zeromean_array);
//     hipFree(d_sig_array);
//     magma_queue_destroy(queue);
// }





// template <typename SDFType>
// void CudaOperation_Base<SDFType>::update_sigmapts(const MatrixXd& covariance, const MatrixXd& mean, int dim_conf, int num_states, MatrixXd& sigmapts){
//     double *covariance_gpu, *mean_gpu, *d_sigmapts;
//     hipMalloc(&covariance_gpu, dim_conf * dim_conf * num_states * sizeof(double));
//     hipMalloc(&mean_gpu, mean.size() * sizeof(double));
//     hipMalloc(&d_sigmapts, _sigmapts_rows * dim_conf * num_states * sizeof(double));

//     hipMemcpy(covariance_gpu, covariance.data(),
//                dim_conf * dim_conf * num_states * sizeof(double), hipMemcpyHostToDevice);
//     hipMemcpy(mean_gpu, mean.data(),
//                mean.size() * sizeof(double), hipMemcpyHostToDevice);

//     // Create 10 streams and corresponding cuSOLVER handles for eigen decomposition
//     const int num_streams = 10;
//     std::vector<hipStream_t> eigen_streams(num_streams);
//     std::vector<hipsolverHandle_t> cusolver_handles(num_streams);
//     for (int i = 0; i < num_streams; i++) {
//         hipStreamCreate(&eigen_streams[i]);
//         hipsolverDnCreate(&cusolver_handles[i]);
//         hipsolverSetStream(cusolver_handles[i], eigen_streams[i]);
//     }

//     // Create a global cuBLAS handle for batched operations (using the default stream)
//     hipblasHandle_t cublas_handle;
//     hipblasCreate(&cublas_handle);

//     // Allocate temporary device memory for each state's operations and record pointers
//     std::vector<double*> d_Pi_array(num_states);
//     std::vector<double*> d_eigen_values_array(num_states);
//     std::vector<double*> d_V_scaled_array(num_states);
//     std::vector<double*> d_sqrtP_array(num_states);
//     std::vector<double*> d_sigmapts_array(num_states);
//     std::vector<double*> d_mean_array(num_states);

//     const double alpha = 1.0, beta = 0.0;
//     for (int state = 0; state < num_states; state++) {
//         // 1. Each state's covariance matrix offset in covariance_gpu
//         double* d_Pi = covariance_gpu + state * dim_conf * dim_conf;
//         d_Pi_array[state] = d_Pi;

//         // 2. Allocate temporary memory for eigen decomposition: eigenvalues and info.
//         double* d_eigen_values;
//         int* d_info;
//         int Lwork = 0;
//         hipMalloc(&d_eigen_values, dim_conf * sizeof(double));
//         hipMalloc(&d_info, sizeof(int));
//         d_eigen_values_array[state] = d_eigen_values;

//         int stream_idx = state % num_streams;  // Assign to one of the 10 streams
//         hipsolverDnDsyevd_bufferSize(cusolver_handles[stream_idx],
//                                     HIPSOLVER_EIG_MODE_VECTOR,
//                                     HIPBLAS_FILL_MODE_LOWER,
//                                     dim_conf,
//                                     d_Pi,
//                                     dim_conf,
//                                     d_eigen_values,
//                                     &Lwork);
//         double* d_work;
//         hipMalloc(&d_work, Lwork * sizeof(double));

//         hipsolverDnDsyevd(cusolver_handles[stream_idx],
//                          HIPSOLVER_EIG_MODE_VECTOR,
//                          HIPBLAS_FILL_MODE_LOWER,
//                          dim_conf,
//                          d_Pi,
//                          dim_conf,
//                          d_eigen_values,
//                          d_work,
//                          Lwork,
//                          d_info);

//         int h_info;
//         hipMemcpy(&h_info, d_info, sizeof(int), hipMemcpyDeviceToHost);
//         if (h_info != 0) {
//             std::cerr << "Eigen decomposition failed for state " << state
//                       << " with info " << h_info << std::endl;
//         }
//         hipFree(d_info);
//         hipFree(d_work);

//         // 3. Apply square root to eigenvalues using a custom kernel on the corresponding stream
//         int threadsPerBlock = 16;
//         int blocks = (dim_conf + threadsPerBlock - 1) / threadsPerBlock;
//         sqrtKernel<<<blocks, threadsPerBlock, 0, eigen_streams[stream_idx]>>>(d_eigen_values, dim_conf);

//         // 4. Allocate memory for ddgmm result (d_V_scaled)
//         double* d_V_scaled;
//         hipMalloc(&d_V_scaled, dim_conf * dim_conf * sizeof(double));
//         d_V_scaled_array[state] = d_V_scaled;

//         // 5. Allocate memory for sqrtP: will compute sqrtP = d_V_scaled * (d_Pi)^T
//         double* d_sqrtP;
//         hipMalloc(&d_sqrtP, dim_conf * dim_conf * sizeof(double));
//         d_sqrtP_array[state] = d_sqrtP;

//         // 6. Each state's sigmapts block stored in d_sigmapts (block size: _sigmapts_rows x dim_conf)
//         double* d_sigmapts_state = d_sigmapts + state * _sigmapts_rows * dim_conf;
//         d_sigmapts_array[state] = d_sigmapts_state;
//         double* d_mi = mean_gpu + 2 * (state + 1) * dim_conf;
//         d_mean_array[state] = d_mi;
//     }
//     // std::cout << "Finished allocating temporary memory" << std::endl;

//     // Synchronize all eigen streams to ensure eigen decomposition and sqrtKernel are completed
//     for (int i = 0; i < num_streams; i++) {
//         hipStreamSynchronize(eigen_streams[i]);
//     }

//     // 4.1 Batched ddgmm: compute d_V_scaled = d_Pi .* diag( sqrt(eigen_values) )
//     int threads = 256;
//     int blocks = (dim_conf * dim_conf * num_states + threads - 1) / threads;
//     batchedDdgmmKernel<<<blocks, threads>>>(d_Pi_array.data(), d_eigen_values_array.data(), d_V_scaled_array.data(), dim_conf, num_states);
//     hipDeviceSynchronize();

//     // 4.2 Batched GEMM: compute sqrtP = d_V_scaled * (d_Pi)^T for each state
//     {
//         // hipblasDgemmBatched requires arrays of pointesanrs for matrices
//         std::vector<const double*> A_array(num_states);  // from d_V_scaled_array
//         std::vector<const double*> B_array(num_states);  // from d_Pi_array (using transpose)
//         std::vector<double*>      C_array(num_states);  // result sqrtP stored in d_sqrtP_array

//         for (int state = 0; state < num_states; state++) {
//             A_array[state] = d_V_scaled_array[state];
//             B_array[state] = d_Pi_array[state];
//             C_array[state] = d_sqrtP_array[state];
//         }
//         hipblasDgemmBatched(cublas_handle,
//                            HIPBLAS_OP_N,   // A is not transposed
//                            HIPBLAS_OP_T,   // B is transposed
//                            dim_conf,      // m
//                            dim_conf,      // n
//                            dim_conf,      // k
//                            &alpha,
//                            A_array.data(), dim_conf,  // Leading dimension of each A
//                            B_array.data(), dim_conf,  // Leading dimension of each B
//                            &beta,
//                            C_array.data(), dim_conf,  // Leading dimension of each C
//                            num_states);

//     }

//     // 4.3 Batched GEMM: compute sigmapts = _zeromean_gpu * (sqrtP)^T for each state
//     {
//         // Assume _zeromean_gpu stores each state's block of size _sigmapts_rows x dim_conf,
//         // and each sqrtP matrix in d_sqrtP_array is of size dim_conf x dim_conf.
//         std::vector<const double*> zeromean_array(num_states);
//         std::vector<const double*> sqrtP_array(num_states);
//         std::vector<double*>       sigmapts_array(num_states);
//         for (int state = 0; state < num_states; state++) {
//             zeromean_array[state] = _zeromean_gpu;
//             sqrtP_array[state] = d_sqrtP_array[state];
//             sigmapts_array[state] = d_sigmapts_array[state];
//         }
//         hipblasDgemmBatched(cublas_handle,
//                            HIPBLAS_OP_N,   // _zeromean_gpu is not transposed
//                            HIPBLAS_OP_T,   // sqrtP is transposed
//                            _sigmapts_rows,  // m
//                            dim_conf,        // n
//                            dim_conf,        // k
//                            &alpha,
//                            zeromean_array.data(), _sigmapts_rows,
//                            sqrtP_array.data(), dim_conf,
//                            &beta,
//                            sigmapts_array.data(), _sigmapts_rows,
//                            num_states);
//     }

//     // 4.4 Batched add mean: add the corresponding mean to each state's sigmapts block
//     int total_elements = _sigmapts_rows * dim_conf * num_states;
//     threads = 256;
//     blocks = (total_elements + threads - 1) / threads;
//     addMeanKernelBatched<<<blocks, threads>>>(d_sigmapts, d_mean_array.data(), _sigmapts_rows, dim_conf, num_states);
//     hipDeviceSynchronize();

//     // Copy the final result from device memory back to host
//     MatrixXd sigma(_sigmapts_rows, dim_conf * num_states);
//     hipMemcpy(sigma.data(), d_sigmapts, _sigmapts_rows * dim_conf * num_states * sizeof(double), hipMemcpyDeviceToHost);
//     sigmapts = sigma;

    // // Free temporary memory for each state and destroy handles and streams
    // for (int state = 0; state < num_states; state++) {
    //     hipFree(d_eigen_values_array[state]);
    //     hipFree(d_V_scaled_array[state]);
    //     hipFree(d_sqrtP_array[state]);
    // }
    // for (int i = 0; i < num_streams; i++) {
    //     hipsolverDnDestroy(cusolver_handles[i]);
    //     hipStreamDestroy(eigen_streams[i]);
    // }
    // // hipblasDestroy(cublas_handle);
    // hipFree(covariance_gpu);
    // hipFree(mean_gpu);
    // hipFree(d_sigmapts);

    // printGPUMemoryInfo();
// }