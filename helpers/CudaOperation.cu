#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <optional>
#include "ngd/NGDFactorizedBaseGH_Cuda.h"
// #include "helpers/CudaOperation.h"

using namespace Eigen;
using GHFunction = std::function<MatrixXd(const VectorXd&)>;

template <typename CostClass>
__global__ void Sigma_function(double* d_sigmapts, double* d_pts, double* mu, int sigmapts_rows, int sigmapts_cols, int res_rows, int res_cols, int type, gvi::NGDFactorizedBaseGH_Cuda<CostClass>* pointer, const CostClass& cost_class){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < sigmapts_rows){
        Eigen::Map<MatrixXd> sigmapts(d_sigmapts, sigmapts_rows, sigmapts_cols);
        Eigen::Map<MatrixXd> pts(d_sigmapts, res_rows, sigmapts_rows*res_cols);
        Eigen::Map<VectorXd> mean(mu, sigmapts_cols);

        double function_value = pointer -> cost_obstacle_planar(sigmapts.row(idx));
        printf("function value: %lf\n", function_value);
        // double function_value = 1;
        // double function_value = pointer -> _function(sigmapts.row(idx), pointer -> _cost_class);

        if (type == 0)
            d_pts[idx] = function_value;
        else if (type == 1){
            for (int i=0; i<sigmapts_cols; i++)
                d_pts[idx*sigmapts_cols + i] = (d_sigmapts[idx + sigmapts_cols * i] - mu[i]) * function_value;
        }
        else{
            for (int i=0; i<sigmapts_cols; i++)
                for (int j=0; j<sigmapts_cols; j++)
                    d_pts[idx*sigmapts_cols *sigmapts_cols+ i*sigmapts_cols +j] = (d_sigmapts[idx*sigmapts_cols + i] - mu[i]) * (d_sigmapts[idx*sigmapts_cols + j] - mu[j]) * function_value;

        }
    }
}

__global__ void obtain_res(double* d_pts, double* d_weights, double* d_result, int sigmapts_rows, int res_rows, int res_cols){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < res_rows && col < res_cols){
        double sum = 0;
        for(int i = 0; i < sigmapts_rows; i++){
            sum += d_pts[i*res_rows*res_cols + col*res_rows + row] * d_weights[i];
        }
        d_result[col*res_rows + row] = sum;
    }
}


namespace gvi{

template <typename CostClass>
void NGDFactorizedBaseGH_Cuda<CostClass>::CudaIntegration(const MatrixXd& sigmapts, const MatrixXd& weights, MatrixXd& results, const MatrixXd& mean, int type, MatrixXd& pts)
{
    double *sigmapts_gpu, *pts_gpu, *weight_gpu, *result_gpu, *mu_gpu;

    NGDFactorizedBaseGH_Cuda<CostClass>* class_gpu;
    hipMalloc(&class_gpu, sizeof(NGDFactorizedBaseGH_Cuda<CostClass>));
    hipMemcpy(class_gpu, this, sizeof(NGDFactorizedBaseGH_Cuda<CostClass>), hipMemcpyHostToDevice);

    // std::cout << sizeof(NGDFactorizedBaseGH<CostClass>) << std::endl;

    hipMalloc(&sigmapts_gpu, sigmapts.size() * sizeof(double));
    hipMalloc(&pts_gpu, sigmapts.rows() * results.size() * sizeof(double));
    hipMalloc(&weight_gpu, sigmapts.rows() * sizeof(double));
    hipMalloc(&result_gpu, results.size() * sizeof(double));
    hipMalloc(&mu_gpu, sigmapts.cols() * sizeof(double));

    hipMemcpy(sigmapts_gpu, sigmapts.data(), sigmapts.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(weight_gpu, weights.data(), sigmapts.rows() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(mu_gpu, mean.data(), sigmapts.cols() * sizeof(double), hipMemcpyHostToDevice);

    // Dimension for the first kernel function
    dim3 blockSize1(3);
    dim3 threadperblock1((sigmapts.rows() + blockSize1.x - 1) / blockSize1.x);

    // std::cout << "Kernel 1" << std::endl;
    // Kernel 1: Obtain the result of function 
    Sigma_function<<<blockSize1, threadperblock1>>>(sigmapts_gpu, pts_gpu, mu_gpu, sigmapts.rows(), sigmapts.cols(), results.rows(), results.cols(), type, class_gpu, this -> _cost_class);
    hipDeviceSynchronize();
    // std::cout << "After Kernel 1" << std::endl;

    hipMemcpy(pts.data(), pts_gpu, sigmapts.rows() * results.size() * sizeof(double), hipMemcpyDeviceToHost);
    // hipMemcpy(pts_gpu, d_pts1, sigma_rows * res_rows * res_cols * sizeof(double), hipMemcpyHostToDevice);
    
    // Dimension for the second kernel function
    dim3 blockSize2(3, 3);
    dim3 threadperblock2((results.cols() + blockSize2.x - 1) / blockSize2.x, (results.rows() + blockSize2.y - 1) / blockSize2.y);

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    obtain_res<<<blockSize2, threadperblock2>>>(pts_gpu, weight_gpu, result_gpu, sigmapts.rows(), results.rows(), results.cols());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(sigmapts_gpu);
    hipFree(pts_gpu);
    hipFree(weight_gpu);
    hipFree(result_gpu);
    
}

// template class NGDFactorizedBaseGH_Cuda<NoneType>;
template class NGDFactorizedBaseGH_Cuda<gpmp2::ObstaclePlanarSDFFactor<gpmp2::PointRobotModel>>;

}