#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <optional>
#include "helpers/CudaOperation.h"

using namespace Eigen;

__global__ void Sigma_function(double* d_sigmapts, double* d_pts, double* mu,
                               int sigmapts_rows, int sigmapts_cols, int res_rows, int res_cols, int type, 
                               gvi::CudaOperation_PlanarPR* pointer, double* d_data){
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < res_rows && col < res_cols*sigmapts_rows){
        int idx = col / res_cols;
        Eigen::Map<MatrixXd> sigmapts(d_sigmapts, sigmapts_rows, sigmapts_cols);

        (pointer->_sdf).data_array_ = d_data;

        double function_value = pointer -> cost_obstacle_planar(sigmapts.row(idx), pointer->_sdf);

        if (type == 0)
            d_pts[idx*res_rows + row] = function_value;
        else if (type == 1)
            d_pts[idx*res_rows + row] = (d_sigmapts[idx + sigmapts_rows * row] - mu[row]) * function_value;
        else{
            int r = col % res_cols;
            d_pts[idx*sigmapts_cols*sigmapts_cols+ r*sigmapts_cols + row] = (d_sigmapts[idx + sigmapts_rows * row] - mu[row]) * (d_sigmapts[idx + sigmapts_rows * r] - mu[r]) * function_value;
        }
    }
}

__global__ void Sigma_function_quad(double* d_sigmapts, double* d_pts, double* mu,
                               int sigmapts_rows, int sigmapts_cols, int res_rows, int res_cols, int type, 
                               gvi::CudaOperation_Quad* pointer, double* d_data){
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < res_rows && col < res_cols*sigmapts_rows){
        int idx = col / res_cols;
        Eigen::Map<MatrixXd> sigmapts(d_sigmapts, sigmapts_rows, sigmapts_cols);

        (pointer->_sdf).data_array_ = d_data;

        double function_value = pointer -> cost_obstacle_planar(sigmapts.row(idx), pointer->_sdf);

        if (type == 0)
            d_pts[idx*res_rows + row] = function_value;
        else if (type == 1)
            d_pts[idx*res_rows + row] = (d_sigmapts[idx + sigmapts_rows * row] - mu[row]) * function_value;
        else{
            int r = col % res_cols;
            d_pts[idx*sigmapts_cols*sigmapts_cols+ r*sigmapts_cols + row] = (d_sigmapts[idx + sigmapts_rows * row] - mu[row]) * (d_sigmapts[idx + sigmapts_rows * r] - mu[r]) * function_value;
        }
    }
}

__global__ void Sigma_function(double* d_sigmapts, double* d_pts, double* mu,
                               int sigmapts_rows, int sigmapts_cols, int res_rows, int res_cols, int type, 
                               gvi::CudaOperation_3dpR* pointer, double* d_data){
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < res_rows && col < res_cols*sigmapts_rows){
        int idx = col / res_cols;
        Eigen::Map<MatrixXd> sigmapts(d_sigmapts, sigmapts_rows, sigmapts_cols);

        (pointer->_sdf).data_array_ = d_data;

        double function_value = pointer -> cost_obstacle_planar(sigmapts.row(idx), pointer->_sdf);

        if (type == 0)
            d_pts[idx*res_rows + row] = function_value;
        else if (type == 1)
            d_pts[idx*res_rows + row] = (d_sigmapts[idx + sigmapts_rows * row] - mu[row]) * function_value;
        else{
            int r = col % res_cols;
            d_pts[idx*sigmapts_cols*sigmapts_cols+ r*sigmapts_cols + row] = (d_sigmapts[idx + sigmapts_rows * row] - mu[row]) * (d_sigmapts[idx + sigmapts_rows * r] - mu[r]) * function_value;
        }
    }
}

__global__ void obtain_res(double* d_pts, double* d_weights, double* d_result, int sigmapts_rows, int res_rows, int res_cols){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < res_rows && col < res_cols){
        double sum = 0;
        for(int i = 0; i < sigmapts_rows; i++){
            sum += d_pts[i*res_rows*res_cols + col*res_rows + row] * d_weights[i];
        }
        d_result[col*res_rows + row] = sum;
    }
}


__global__ void cost_function(double* d_sigmapts, double* d_pts, int sigmapts_rows, int sigmapts_cols, 
                                int n_states, gvi::CudaOperation_PlanarPR* pointer, double* d_data){
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < sigmapts_rows && col < n_states){
        Eigen::Map<MatrixXd> sigmapts(d_sigmapts + col*sigmapts_rows*sigmapts_cols, sigmapts_rows, sigmapts_cols);

        (pointer->_sdf).data_array_ = d_data;

        double function_value = pointer -> cost_obstacle_planar(sigmapts.row(row), pointer->_sdf);

        d_pts[col*sigmapts_rows + row] = function_value;
    }
}

__global__ void cost_function(double* d_sigmapts, double* d_pts, int sigmapts_rows, int sigmapts_cols, 
                                int n_states, gvi::CudaOperation_Quad* pointer, double* d_data){
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < sigmapts_rows && col < n_states){
        Eigen::Map<MatrixXd> sigmapts(d_sigmapts + col*sigmapts_rows*sigmapts_cols, sigmapts_rows, sigmapts_cols);

        (pointer->_sdf).data_array_ = d_data;

        double function_value = pointer -> cost_obstacle_planar(sigmapts.row(row), pointer->_sdf);

        d_pts[col*sigmapts_rows + row] = function_value;
    }
}

__global__ void cost_function(double* d_sigmapts, double* d_pts, int sigmapts_rows, int sigmapts_cols, 
                                int n_states, gvi::CudaOperation_3dpR* pointer, double* d_data){
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < sigmapts_rows && col < n_states){
        Eigen::Map<MatrixXd> sigmapts(d_sigmapts + col*sigmapts_rows*sigmapts_cols, sigmapts_rows, sigmapts_cols);

        (pointer->_sdf).data_array_ = d_data;

        double function_value = pointer -> cost_obstacle_planar(sigmapts.row(row), pointer->_sdf);

        d_pts[col*sigmapts_rows + row] = function_value;
    }
}

__global__ void dmu_function(double* d_sigmapts, double* d_mu, double* d_pts, double* d_vec, int sigmapts_rows, int dim_state, int n_states){
// __global__ void dmu_function(int sigmapts_rows, int dim_state, int n_states){
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < sigmapts_rows && col < n_states*dim_state){
        int idx = col / dim_state;
        int r = col % dim_state;

        d_vec[col*sigmapts_rows + row] = (d_sigmapts[col*sigmapts_rows + row] - d_mu[idx*dim_state + r]) * d_pts[idx * sigmapts_rows + row];
    }
}

__global__ void ddmu_function(double* d_sigmapts, double* d_mu, double* d_pts, double* d_vec, int sigmapts_rows, int sigmapts_cols, int n_states){
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < sigmapts_rows*sigmapts_cols && col < n_states*sigmapts_cols){
        int idx_x = col / sigmapts_cols; // Use which mu
        int idx_y = row / sigmapts_cols; // use which sigma
        int mat_x = col % sigmapts_cols; // 
        int mat_y = row % sigmapts_cols;

        d_vec[col*sigmapts_rows*sigmapts_cols + row] = (d_sigmapts[(idx_x*sigmapts_cols + mat_x) * sigmapts_rows + idx_y] - d_mu[idx_x*sigmapts_cols + mat_x]) 
                                                     * (d_sigmapts[(idx_x*sigmapts_cols + mat_y) * sigmapts_rows + idx_y] - d_mu[idx_x*sigmapts_cols + mat_y]) 
                                                     * d_pts[idx_x * sigmapts_rows + idx_y];
    }
}

__global__ void obtain_cost(double* d_pts, double* d_weights, double* d_result, int sigmapts_rows, int n_states){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n_states){
        double sum = 0;
        for(int i = 0; i < sigmapts_rows; i++){
            sum += d_pts[idx*sigmapts_rows + i]* d_weights[i];
        }
        d_result[idx] = sum;
    }
}

__global__ void obtain_dmu(double* d_vec, double* d_weights, double* d_result, int sigmapts_rows, int res_cols){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < res_cols){
        double sum = 0;
        for(int i = 0; i < sigmapts_rows; i++){
            sum += d_vec[idx*sigmapts_rows + i]* d_weights[i];
        }
        d_result[idx] = sum;
    }
}

__global__ void obtain_ddmu(double* d_vec, double* d_weights, double* d_result, int sigmapts_rows, int res_rows, int res_cols){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < res_rows && col < res_cols){
        double sum = 0;
        for(int i = 0; i < sigmapts_rows; i++){
            sum += d_vec[col*sigmapts_rows*res_rows + row + i*res_rows] * d_weights[i];
        }
        d_result[col*res_rows + row] = sum;
    }
}


namespace gvi{

void CudaOperation_PlanarPR::CudaIntegration(const MatrixXd& sigmapts, const MatrixXd& weights, MatrixXd& results, const MatrixXd& mean, int type)
{
    double *sigmapts_gpu, *mu_gpu, *pts_gpu, *result_gpu;
    int n_balls = 1;

    hipMalloc(&sigmapts_gpu, sigmapts.size() * sizeof(double));
    hipMalloc(&mu_gpu, sigmapts.cols() * sizeof(double));
    hipMalloc(&pts_gpu, sigmapts.rows() * results.size() * sizeof(double));
    hipMalloc(&result_gpu, results.size() * sizeof(double));

    hipMemcpy(sigmapts_gpu, sigmapts.data(), sigmapts.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(mu_gpu, mean.data(), sigmapts.cols() * sizeof(double), hipMemcpyHostToDevice);

    // Kernel 1: Obtain the result of function 
    dim3 blockSize1(16, 16);
    dim3 threadperblock1((results.cols()*sigmapts.rows() + blockSize1.x - 1) / blockSize1.x, (results.rows() + blockSize1.y - 1) / blockSize1.y);

    Sigma_function<<<blockSize1, threadperblock1>>>(sigmapts_gpu, pts_gpu, mu_gpu, sigmapts.rows(), sigmapts.cols(), results.rows(), results.cols(), type, _class_gpu, _data_gpu);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(sigmapts_gpu);
    hipFree(mu_gpu);
    

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 blockSize2(16, 16);
    dim3 threadperblock2((results.cols() + blockSize2.x - 1) / blockSize2.x, (results.rows() + blockSize2.y - 1) / blockSize2.y);

    obtain_res<<<blockSize2, threadperblock2>>>(pts_gpu, _weight_gpu, result_gpu, sigmapts.rows(), results.rows(), results.cols());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(pts_gpu);
    hipFree(result_gpu);
}

void CudaOperation_PlanarPR::costIntegration(const MatrixXd& sigmapts, VectorXd& results, const int sigmapts_cols){
    double *result_gpu;
    hipMalloc(&result_gpu, results.size() * sizeof(double));

    hipMemcpy(_sigmapts_gpu, sigmapts.data(), sigmapts.size() * sizeof(double), hipMemcpyHostToDevice);

    // Kernel 1: Obtain the result of function 
    dim3 blockSize1(16, 16);
    dim3 threadperblock1((results.size() + blockSize1.x - 1) / blockSize1.x, (sigmapts.rows() + blockSize1.y - 1) / blockSize1.y);

    cost_function<<<blockSize1, threadperblock1>>>(_sigmapts_gpu, _func_value_gpu, sigmapts.rows(), sigmapts_cols, results.size(), _class_gpu, _data_gpu);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 blockSize2(16);
    dim3 threadperblock2((results.size() + blockSize2.x - 1) / blockSize2.x);

    obtain_cost<<<blockSize2, threadperblock2>>>(_func_value_gpu, _weight_gpu, result_gpu, sigmapts.rows(), results.size());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(result_gpu);
}

void CudaOperation_PlanarPR::dmuIntegration(const MatrixXd& sigmapts, const MatrixXd& mu, VectorXd& results, const int sigmapts_cols){

    double *vec_gpu, *result_gpu;
    hipMalloc(&vec_gpu, sigmapts.rows() * results.size() * sizeof(double));
    hipMalloc(&result_gpu, results.size() * sizeof(double));
    hipMalloc(&_mu_gpu, mu.size() * sizeof(double));

    hipMemcpy(_mu_gpu, mu.data(), mu.size() * sizeof(double), hipMemcpyHostToDevice);

    // Kernel 1: Obtain the result of function 
    dim3 blockSize1(16, 16);
    dim3 threadperblock1((results.size() + blockSize1.x - 1) / blockSize1.x, (_sigmapts_rows + blockSize1.y - 1) / blockSize1.y);

    dmu_function<<<blockSize1, threadperblock1>>>(_sigmapts_gpu, _mu_gpu, _func_value_gpu, vec_gpu, _sigmapts_rows, _dim_state, _n_states);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 blockSize2(16);
    dim3 threadperblock2((results.size() + blockSize2.x - 1) / blockSize2.x);

    obtain_dmu<<<blockSize2, threadperblock2>>>(vec_gpu, _weight_gpu, result_gpu, sigmapts.rows(), results.size());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(vec_gpu);
    hipFree(result_gpu);
}

void CudaOperation_PlanarPR::ddmuIntegration(MatrixXd& results){
    // Reuse the sigmapts and pts passed into gpu before
    double *vec_gpu, *result_gpu;
    hipMalloc(&vec_gpu, _sigmapts_rows * results.size() * sizeof(double));
    hipMalloc(&result_gpu, results.size() * sizeof(double));

    // Kernel 1: Obtain the result of function 
    dim3 blockSize1(16, 16);
    dim3 threadperblock1((results.cols() + blockSize1.x - 1) / blockSize1.x, (_sigmapts_rows * results.rows() + blockSize1.y - 1) / blockSize1.y);

    ddmu_function<<<blockSize1, threadperblock1>>>(_sigmapts_gpu, _mu_gpu, _func_value_gpu, vec_gpu, _sigmapts_rows, _dim_state, _n_states);
    hipDeviceSynchronize();

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 blockSize2(16, 16);
    dim3 threadperblock2((results.cols() + blockSize2.x - 1) / blockSize2.x, (results.rows() + blockSize2.y - 1) / blockSize2.y);

    obtain_ddmu<<<blockSize2, threadperblock2>>>(vec_gpu, _weight_gpu, result_gpu, _sigmapts_rows, results.rows(), results.cols());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(vec_gpu);
    hipFree(result_gpu);
    hipFree(_mu_gpu);
}


void CudaOperation_Quad::CudaIntegration(const MatrixXd& sigmapts, const MatrixXd& weights, MatrixXd& results, const MatrixXd& mean, int type)
{
    double *sigmapts_gpu, *pts_gpu, *result_gpu, *mu_gpu;
    int n_balls = 5;

    hipMalloc(&sigmapts_gpu, sigmapts.size() * sizeof(double));
    hipMalloc(&pts_gpu, sigmapts.rows() * results.size() * sizeof(double));
    hipMalloc(&result_gpu, results.size() * sizeof(double));
    hipMalloc(&mu_gpu, sigmapts.cols() * sizeof(double));

    hipMemcpy(sigmapts_gpu, sigmapts.data(), sigmapts.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(mu_gpu, mean.data(), sigmapts.cols() * sizeof(double), hipMemcpyHostToDevice);

    // Kernel 1: Obtain the result of function 
    dim3 blockSize1(16, 16);
    dim3 threadperblock1((results.cols()*sigmapts.rows() + blockSize1.x - 1) / blockSize1.x, (results.rows() + blockSize1.y - 1) / blockSize1.y);

    Sigma_function_quad<<<blockSize1, threadperblock1>>>(sigmapts_gpu, pts_gpu, mu_gpu, sigmapts.rows(), sigmapts.cols(), results.rows(), results.cols(), type, _class_gpu, _data_gpu);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(sigmapts_gpu);
    hipFree(mu_gpu);

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 blockSize2(16, 16);
    dim3 threadperblock2((results.cols() + blockSize2.x - 1) / blockSize2.x, (results.rows() + blockSize2.y - 1) / blockSize2.y);

    obtain_res<<<blockSize2, threadperblock2>>>(pts_gpu, _weight_gpu, result_gpu, sigmapts.rows(), results.rows(), results.cols());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(pts_gpu);
    hipFree(result_gpu);
}

void CudaOperation_Quad::costIntegration(const MatrixXd& sigmapts, VectorXd& results, const int sigmapts_cols){
    double *result_gpu;
    hipMalloc(&result_gpu, results.size() * sizeof(double));

    hipMemcpy(_sigmapts_gpu, sigmapts.data(), sigmapts.size() * sizeof(double), hipMemcpyHostToDevice);

    // Kernel 1: Obtain the result of function 
    dim3 blockSize1(16, 16);
    dim3 threadperblock1((results.size() + blockSize1.x - 1) / blockSize1.x, (sigmapts.rows() + blockSize1.y - 1) / blockSize1.y);

    cost_function<<<blockSize1, threadperblock1>>>(_sigmapts_gpu, _func_value_gpu, sigmapts.rows(), sigmapts_cols, results.size(), _class_gpu, _data_gpu);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 blockSize2(16);
    dim3 threadperblock2((results.size() + blockSize2.x - 1) / blockSize2.x);

    obtain_cost<<<blockSize2, threadperblock2>>>(_func_value_gpu, _weight_gpu, result_gpu, sigmapts.rows(), results.size());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    // hipFree(_func_value_gpu);
    hipFree(result_gpu);
}

void CudaOperation_Quad::dmuIntegration(const MatrixXd& sigmapts, const MatrixXd& mu, VectorXd& results, const int sigmapts_cols){

    double *vec_gpu, *result_gpu;
    hipMalloc(&vec_gpu, sigmapts.rows() * results.size() * sizeof(double));
    hipMalloc(&result_gpu, results.size() * sizeof(double));
    hipMalloc(&_mu_gpu, mu.size() * sizeof(double));

    hipMemcpy(_mu_gpu, mu.data(), mu.size() * sizeof(double), hipMemcpyHostToDevice);

    // Kernel 1: Obtain the result of function 
    dim3 blockSize1(16, 16);
    dim3 threadperblock1((results.size() + blockSize1.x - 1) / blockSize1.x, (_sigmapts_rows + blockSize1.y - 1) / blockSize1.y);

    dmu_function<<<blockSize1, threadperblock1>>>(_sigmapts_gpu, _mu_gpu, _func_value_gpu, vec_gpu, _sigmapts_rows, _dim_state, _n_states);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 blockSize2(16);
    dim3 threadperblock2((results.size() + blockSize2.x - 1) / blockSize2.x);

    obtain_dmu<<<blockSize2, threadperblock2>>>(vec_gpu, _weight_gpu, result_gpu, sigmapts.rows(), results.size());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(vec_gpu);
    hipFree(result_gpu);
}

void CudaOperation_Quad::ddmuIntegration(MatrixXd& results){
    // Reuse the sigmapts and pts passed into gpu before
    double *vec_gpu, *result_gpu;
    hipMalloc(&vec_gpu, _sigmapts_rows * results.size() * sizeof(double));
    hipMalloc(&result_gpu, results.size() * sizeof(double));

    // Kernel 1: Obtain the result of function 
    dim3 blockSize1(16, 16);
    dim3 threadperblock1((results.cols() + blockSize1.x - 1) / blockSize1.x, (_sigmapts_rows * results.rows() + blockSize1.y - 1) / blockSize1.y);

    ddmu_function<<<blockSize1, threadperblock1>>>(_sigmapts_gpu, _mu_gpu, _func_value_gpu, vec_gpu, _sigmapts_rows, _dim_state, _n_states);
    hipDeviceSynchronize();

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 blockSize2(16, 16);
    dim3 threadperblock2((results.cols() + blockSize2.x - 1) / blockSize2.x, (results.rows() + blockSize2.y - 1) / blockSize2.y);

    obtain_ddmu<<<blockSize2, threadperblock2>>>(vec_gpu, _weight_gpu, result_gpu, _sigmapts_rows, results.rows(), results.cols());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(vec_gpu);
    hipFree(result_gpu);
    hipFree(_mu_gpu);
}


void CudaOperation_3dpR::CudaIntegration(const MatrixXd& sigmapts, const MatrixXd& weights, MatrixXd& results, const MatrixXd& mean, int type)
{
    double *sigmapts_gpu, *mu_gpu, *pts_gpu, *result_gpu;
    int n_balls = 1;

    hipMalloc(&sigmapts_gpu, sigmapts.size() * sizeof(double));
    hipMalloc(&mu_gpu, sigmapts.cols() * sizeof(double));
    hipMalloc(&pts_gpu, sigmapts.rows() * results.size() * sizeof(double));
    hipMalloc(&result_gpu, results.size() * sizeof(double));

    hipMemcpy(sigmapts_gpu, sigmapts.data(), sigmapts.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(mu_gpu, mean.data(), sigmapts.cols() * sizeof(double), hipMemcpyHostToDevice);

    // Kernel 1: Obtain the result of function 
    dim3 blockSize1(16, 16);
    dim3 threadperblock1((results.cols()*sigmapts.rows() + blockSize1.x - 1) / blockSize1.x, (results.rows() + blockSize1.y - 1) / blockSize1.y);

    Sigma_function<<<blockSize1, threadperblock1>>>(sigmapts_gpu, pts_gpu, mu_gpu, sigmapts.rows(), sigmapts.cols(), results.rows(), results.cols(), type, _class_gpu, _data_gpu);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(sigmapts_gpu);
    hipFree(mu_gpu);
    

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 blockSize2(16, 16);
    dim3 threadperblock2((results.cols() + blockSize2.x - 1) / blockSize2.x, (results.rows() + blockSize2.y - 1) / blockSize2.y);

    obtain_res<<<blockSize2, threadperblock2>>>(pts_gpu, _weight_gpu, result_gpu, sigmapts.rows(), results.rows(), results.cols());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(pts_gpu);
    hipFree(result_gpu);
}

void CudaOperation_3dpR::costIntegration(const MatrixXd& sigmapts, VectorXd& results, const int sigmapts_cols){
    double *result_gpu;

    hipMalloc(&result_gpu, results.size() * sizeof(double));

    hipMemcpy(_sigmapts_gpu, sigmapts.data(), sigmapts.size() * sizeof(double), hipMemcpyHostToDevice);

    // Kernel 1: Obtain the result of function 
    dim3 blockSize1(16, 16);
    dim3 threadperblock1((results.size() + blockSize1.x - 1) / blockSize1.x, (sigmapts.rows() + blockSize1.y - 1) / blockSize1.y);

    cost_function<<<blockSize1, threadperblock1>>>(_sigmapts_gpu, _func_value_gpu, sigmapts.rows(), sigmapts_cols, results.size(), _class_gpu, _data_gpu);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 blockSize2(16);
    dim3 threadperblock2((results.size() + blockSize2.x - 1) / blockSize2.x);

    obtain_cost<<<blockSize2, threadperblock2>>>(_func_value_gpu, _weight_gpu, result_gpu, sigmapts.rows(), results.size());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(result_gpu);
}

void CudaOperation_3dpR::dmuIntegration(const MatrixXd& sigmapts, const MatrixXd& mu, VectorXd& results, const int sigmapts_cols){

    double *vec_gpu, *result_gpu;
    hipMalloc(&vec_gpu, sigmapts.rows() * results.size() * sizeof(double));
    hipMalloc(&result_gpu, results.size() * sizeof(double));
    hipMalloc(&_mu_gpu, mu.size() * sizeof(double));

    hipMemcpy(_mu_gpu, mu.data(), mu.size() * sizeof(double), hipMemcpyHostToDevice);

    // Kernel 1: Obtain the result of function 
    dim3 blockSize1(16, 16);
    dim3 threadperblock1((results.size() + blockSize1.x - 1) / blockSize1.x, (_sigmapts_rows + blockSize1.y - 1) / blockSize1.y);

    dmu_function<<<blockSize1, threadperblock1>>>(_sigmapts_gpu, _mu_gpu, _func_value_gpu, vec_gpu, _sigmapts_rows, _dim_state, _n_states);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 blockSize2(16);
    dim3 threadperblock2((results.size() + blockSize2.x - 1) / blockSize2.x);

    obtain_dmu<<<blockSize2, threadperblock2>>>(vec_gpu, _weight_gpu, result_gpu, sigmapts.rows(), results.size());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(vec_gpu);
    hipFree(result_gpu);
}

void CudaOperation_3dpR::ddmuIntegration(MatrixXd& results){
    // Reuse the sigmapts and pts passed into gpu before
    double *vec_gpu, *result_gpu;
    hipMalloc(&vec_gpu, _sigmapts_rows * results.size() * sizeof(double));
    hipMalloc(&result_gpu, results.size() * sizeof(double));

    // Kernel 1: Obtain the result of function 
    dim3 blockSize1(64, 64);
    dim3 threadperblock1((results.cols() + blockSize1.x - 1) / blockSize1.x, (_sigmapts_rows * results.rows() + blockSize1.y - 1) / blockSize1.y);

    ddmu_function<<<blockSize1, threadperblock1>>>(_sigmapts_gpu, _mu_gpu, _func_value_gpu, vec_gpu, _sigmapts_rows, _dim_state, _n_states);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 blockSize2(64, 64);
    dim3 threadperblock2((results.cols() + blockSize2.x - 1) / blockSize2.x, (results.rows() + blockSize2.y - 1) / blockSize2.y);

    obtain_ddmu<<<blockSize2, threadperblock2>>>(vec_gpu, _weight_gpu, result_gpu, _sigmapts_rows, results.rows(), results.cols());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(vec_gpu);
    hipFree(result_gpu);
    hipFree(_mu_gpu);
}

}





// void CudaOperation_PlanarPR::funcValueIntegration(const MatrixXd& sigmapts, VectorXd& results, const int sigmapts_cols){

//     double *result_gpu;
//     hipMalloc(&result_gpu, results.size() * sizeof(double));

//     hipMemcpy(_sigmapts_gpu, sigmapts.data(), sigmapts.size() * sizeof(double), hipMemcpyHostToDevice);

//     // Kernel 1: Obtain the result of function 
//     dim3 blockSize1(16, 16);
//     dim3 threadperblock1((_n_states + blockSize1.x - 1) / blockSize1.x, (sigmapts.rows() + blockSize1.y - 1) / blockSize1.y);

//     cost_function<<<blockSize1, threadperblock1>>>(_sigmapts_gpu, _func_value_gpu, sigmapts.rows(), sigmapts_cols, _n_states, _class_gpu, _data_gpu);
//     hipDeviceSynchronize();

//     hipError_t err = hipGetLastError();
//     if (err != hipSuccess) {
//         printf("CUDA kernel error: %s\n", hipGetErrorString(err));
//     }

//     // Kernel 2: Obtain the result by multiplying the pts and the weights
//     dim3 blockSize2(16);
//     dim3 threadperblock2((results.size() + blockSize2.x - 1) / blockSize2.x);

//     obtain_cost<<<blockSize2, threadperblock2>>>(_func_value_gpu, _weight_gpu, result_gpu, sigmapts.rows(), results.size());
//     hipDeviceSynchronize();
//     hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

//     err = hipGetLastError();
//     if (err != hipSuccess) {
//         printf("CUDA kernel error: %s\n", hipGetErrorString(err));
//     }

//     hipFree(result_gpu);
// }