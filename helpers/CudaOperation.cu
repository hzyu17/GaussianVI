#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <optional>
#include "ngd/NGDFactorizedBaseGH_Cuda.h"
#include <gpmp2/obstacle/ObstaclePlanarSDFFactor.h>
// #include "helpers/CudaOperation.h"

using namespace Eigen;
using GHFunction = std::function<MatrixXd(const VectorXd&)>;

template <typename CostClass>
__global__ void Sigma_function(double* d_sigmapts, double* d_pts, double* mu,
                               int sigmapts_rows, int sigmapts_cols, int res_rows, int res_cols, int type, 
                               gvi::NGDFactorizedBaseGH_Cuda<CostClass>* pointer, gvi::PlanarSDF* sdf, double* d_data){
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < sigmapts_rows){
        Eigen::Map<MatrixXd> sigmapts(d_sigmapts, sigmapts_rows, sigmapts_cols);
        Eigen::Map<MatrixXd> pts(d_sigmapts, res_rows, sigmapts_rows*res_cols);
        Eigen::Map<MatrixXd> data_map(d_data, sdf->field_rows_, sdf->field_cols_);
        Eigen::Map<VectorXd> mean(mu, sigmapts_cols);
        // printf("idx = %d(%lf, %lf)\n", idx, sigmapts.row(idx)(0), sigmapts.row(idx)(1));
        // sdf -> origin_ = origin;
        // sdf -> data_ = data;
        // printf("Origin = (%lf, %lf)\n", sdf->origin_(0), sdf->origin_(1));
        sdf -> data_array = d_data;

        // printf("data[%d] = %lf\n", idx, sdf -> data_array[idx]);
        double function_value = pointer -> cost_obstacle_planar(sigmapts.row(idx), *sdf);

        // printf("idx = %d, function value: %lf\n", idx, function_value);
        // double function_value = 1;
        // double function_value = pointer -> _function(sigmapts.row(idx), pointer -> _cost_class);

        if (type == 0)
            d_pts[idx] = function_value;
        else if (type == 1){
            for (int i=0; i<sigmapts_cols; i++)
                d_pts[idx*sigmapts_cols + i] = (d_sigmapts[idx + sigmapts_rows * i] - mu[i]) * function_value;
        }
        else{
            for (int i=0; i<sigmapts_cols; i++)
                for (int j=0; j<sigmapts_cols; j++)
                    d_pts[idx*sigmapts_cols *sigmapts_cols+ j*sigmapts_cols + i] = (d_sigmapts[idx + sigmapts_rows * i] - mu[i]) * (d_sigmapts[idx + sigmapts_rows * j] - mu[j]) * function_value;

        }
    }
}

__global__ void obtain_res(double* d_pts, double* d_weights, double* d_result, int sigmapts_rows, int res_rows, int res_cols){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < res_rows && col < res_cols){
        double sum = 0;
        for(int i = 0; i < sigmapts_rows; i++){
            sum += d_pts[i*res_rows*res_cols + col*res_rows + row] * d_weights[i];
        }
        d_result[col*res_rows + row] = sum;
    }
}


namespace gvi{

template <typename CostClass>
void NGDFactorizedBaseGH_Cuda<CostClass>::CudaIntegration(const MatrixXd& sigmapts, const MatrixXd& weights, MatrixXd& results, const MatrixXd& mean, int type, MatrixXd& pts)
{
    double *sigmapts_gpu, *pts_gpu, *weight_gpu, *result_gpu, *mu_gpu, *data_gpu;
    // std::cout << "Using cuda" << std::endl;
    // MatrixIO _m_io;
    // std::string field_file = source_root + "/maps/2dpR/map2/field_multiobs_map2.csv";
    // MatrixXd field = _m_io.load_csv(field_file);      

    // Vector2d origin;
    // origin.setZero();
    // origin << -20.0, -10.0;

    // double cell_size = 0.1;

    // PlanarSDF sdf(origin, cell_size, field);

    PlanarSDF* sdf_gpu;
    hipMalloc(&sdf_gpu, sizeof(PlanarSDF));
    hipMemcpy(sdf_gpu, &_sdf, sizeof(PlanarSDF), hipMemcpyHostToDevice);

    NGDFactorizedBaseGH_Cuda<CostClass>* class_gpu;
    hipMalloc(&class_gpu, sizeof(NGDFactorizedBaseGH_Cuda<CostClass>));
    hipMemcpy(class_gpu, this, sizeof(NGDFactorizedBaseGH_Cuda<CostClass>), hipMemcpyHostToDevice);


    hipMalloc(&sigmapts_gpu, sigmapts.size() * sizeof(double));
    hipMalloc(&pts_gpu, sigmapts.rows() * results.size() * sizeof(double));
    hipMalloc(&weight_gpu, sigmapts.rows() * sizeof(double));
    hipMalloc(&result_gpu, results.size() * sizeof(double));
    hipMalloc(&mu_gpu, sigmapts.cols() * sizeof(double));
    hipMalloc(&data_gpu, _sdf.data_.size() * sizeof(double));


    hipMemcpy(sigmapts_gpu, sigmapts.data(), sigmapts.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(weight_gpu, weights.data(), sigmapts.rows() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(mu_gpu, mean.data(), sigmapts.cols() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(data_gpu, _sdf.data_.data(), _sdf.data_.size() * sizeof(double), hipMemcpyHostToDevice);

    // Dimension for the first kernel function
    dim3 blockSize1(3);
    dim3 threadperblock1((sigmapts.rows() + blockSize1.x - 1) / blockSize1.x);

    // Kernel 1: Obtain the result of function 
    Sigma_function<<<blockSize1, threadperblock1>>>(sigmapts_gpu, pts_gpu, mu_gpu, sigmapts.rows(), sigmapts.cols(), results.rows(), results.cols(), type, class_gpu, sdf_gpu, data_gpu);
    hipDeviceSynchronize();

    hipMemcpy(pts.data(), pts_gpu, sigmapts.rows() * results.size() * sizeof(double), hipMemcpyDeviceToHost);
    // hipMemcpy(pts_gpu, d_pts1, sigma_rows * res_rows * res_cols * sizeof(double), hipMemcpyHostToDevice);
    
    // Dimension for the second kernel function
    dim3 blockSize2(3, 3);
    dim3 threadperblock2((results.cols() + blockSize2.x - 1) / blockSize2.x, (results.rows() + blockSize2.y - 1) / blockSize2.y);

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    obtain_res<<<blockSize2, threadperblock2>>>(pts_gpu, weight_gpu, result_gpu, sigmapts.rows(), results.rows(), results.cols());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(sigmapts_gpu);
    hipFree(pts_gpu);
    hipFree(weight_gpu);
    hipFree(result_gpu);
    
}

// template class NGDFactorizedBaseGH_Cuda<NoneType>;
template class NGDFactorizedBaseGH_Cuda<gpmp2::ObstaclePlanarSDFFactor<gpmp2::PointRobotModel>>;

}