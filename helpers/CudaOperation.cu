#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <optional>
#include "helpers/CudaOperation.h"

using namespace Eigen;

__global__ void Sigma_function(double* d_sigmapts, double* d_pts, double* mu,
                               int sigmapts_rows, int sigmapts_cols, int res_rows, int res_cols, int type, 
                               gvi::CudaOperation* pointer, double* d_data){
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < res_rows && col < res_cols*sigmapts_rows){
        int idx = col / res_cols;
        Eigen::Map<MatrixXd> sigmapts(d_sigmapts, sigmapts_rows, sigmapts_cols);

        (pointer->_sdf).data_array = d_data;

        double function_value = pointer -> cost_obstacle_planar(sigmapts.row(idx), pointer->_sdf);

        if (type == 0)
            d_pts[idx*res_rows + row] = function_value;
        else if (type == 1)
            d_pts[idx*res_rows + row] = (d_sigmapts[idx + sigmapts_rows * row] - mu[row]) * function_value;
        else{
            int r = col % res_cols;
            d_pts[idx*sigmapts_cols*sigmapts_cols+ r*sigmapts_cols + row] = (d_sigmapts[idx + sigmapts_rows * row] - mu[row]) * (d_sigmapts[idx + sigmapts_rows * r] - mu[r]) * function_value;
        }
    }
}

__global__ void Sigma_function(double* d_sigmapts, double* d_pts, double* mu,
                               int sigmapts_rows, int sigmapts_cols, int res_rows, int res_cols, int type, 
                               gvi::CudaOperation_Quad* pointer, double* d_data){
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < res_rows && col < res_cols*sigmapts_rows){
        int idx = col / res_cols;
        Eigen::Map<MatrixXd> sigmapts(d_sigmapts, sigmapts_rows, sigmapts_cols);

        (pointer->_sdf).data_array = d_data;

        double function_value = pointer -> cost_obstacle_planar(sigmapts.row(idx), pointer->_sdf);

        if (type == 0)
            d_pts[idx*res_rows + row] = function_value;
        else if (type == 1)
            d_pts[idx*res_rows + row] = (d_sigmapts[idx + sigmapts_rows * row] - mu[row]) * function_value;
        else{
            int r = col % res_cols;
            d_pts[idx*sigmapts_cols*sigmapts_cols+ r*sigmapts_cols + row] = (d_sigmapts[idx + sigmapts_rows * row] - mu[row]) * (d_sigmapts[idx + sigmapts_rows * r] - mu[r]) * function_value;
        }
    }
}

__global__ void obtain_res(double* d_pts, double* d_weights, double* d_result, int sigmapts_rows, int res_rows, int res_cols){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < res_rows && col < res_cols){
        double sum = 0;
        for(int i = 0; i < sigmapts_rows; i++){
            sum += d_pts[i*res_rows*res_cols + col*res_rows + row] * d_weights[i];
        }
        d_result[col*res_rows + row] = sum;
    }
}


namespace gvi{

void CudaOperation::Cuda_init(const MatrixXd& weights)
{
    hipMalloc(&_weight_gpu, weights.size() * sizeof(double));
    hipMalloc(&_data_gpu, _sdf.data_.size() * sizeof(double));
    hipMalloc(&_class_gpu, sizeof(CudaOperation));

    hipMemcpy(_weight_gpu, weights.data(), weights.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(_data_gpu, _sdf.data_.data(), _sdf.data_.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(_class_gpu, this, sizeof(CudaOperation), hipMemcpyHostToDevice);
}

void CudaOperation::CudaIntegration(const MatrixXd& sigmapts, const MatrixXd& weights, MatrixXd& results, const MatrixXd& mean, int type)
{
    double *sigmapts_gpu, *mu_gpu, *pts_gpu, *result_gpu;
    int n_balls = 1;

    hipMalloc(&sigmapts_gpu, sigmapts.size() * sizeof(double));
    hipMalloc(&mu_gpu, sigmapts.cols() * sizeof(double));
    hipMalloc(&pts_gpu, sigmapts.rows() * results.size() * sizeof(double));
    hipMalloc(&result_gpu, results.size() * sizeof(double));

    hipMemcpy(sigmapts_gpu, sigmapts.data(), sigmapts.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(mu_gpu, mean.data(), sigmapts.cols() * sizeof(double), hipMemcpyHostToDevice);

    // Kernel 1: Obtain the result of function 
    dim3 blockSize1(16, 16);
    dim3 threadperblock1((results.cols()*sigmapts.rows() + blockSize1.x - 1) / blockSize1.x, (results.rows() + blockSize1.y - 1) / blockSize1.y);

    Sigma_function<<<blockSize1, threadperblock1>>>(sigmapts_gpu, pts_gpu, mu_gpu, sigmapts.rows(), sigmapts.cols(), results.rows(), results.cols(), type, _class_gpu, _data_gpu);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(sigmapts_gpu);
    hipFree(mu_gpu);
    

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 blockSize2(16, 16);
    dim3 threadperblock2((results.cols() + blockSize2.x - 1) / blockSize2.x, (results.rows() + blockSize2.y - 1) / blockSize2.y);

    obtain_res<<<blockSize2, threadperblock2>>>(pts_gpu, _weight_gpu, result_gpu, sigmapts.rows(), results.rows(), results.cols());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(pts_gpu);
    hipFree(result_gpu);
}

void CudaOperation::Cuda_free()
{
    hipFree(_weight_gpu);
    hipFree(_data_gpu);
    hipFree(_class_gpu);
}

void CudaOperation_Quad::Cuda_init(const MatrixXd& weights)
{
    hipMalloc(&_weight_gpu, weights.size() * sizeof(double));
    hipMalloc(&_data_gpu, _sdf.data_.size() * sizeof(double));
    hipMalloc(&_class_gpu, sizeof(CudaOperation_Quad));

    hipMemcpy(_weight_gpu, weights.data(), weights.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(_data_gpu, _sdf.data_.data(), _sdf.data_.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(_class_gpu, this, sizeof(CudaOperation_Quad), hipMemcpyHostToDevice);
}

void CudaOperation_Quad::CudaIntegration(const MatrixXd& sigmapts, const MatrixXd& weights, MatrixXd& results, const MatrixXd& mean, int type)
{
    double *sigmapts_gpu, *pts_gpu, *result_gpu, *mu_gpu;
    int n_balls = 5;

    hipMalloc(&sigmapts_gpu, sigmapts.size() * sizeof(double));
    hipMalloc(&pts_gpu, sigmapts.rows() * results.size() * sizeof(double));
    hipMalloc(&result_gpu, results.size() * sizeof(double));
    hipMalloc(&mu_gpu, sigmapts.cols() * sizeof(double));

    hipMemcpy(sigmapts_gpu, sigmapts.data(), sigmapts.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(mu_gpu, mean.data(), sigmapts.cols() * sizeof(double), hipMemcpyHostToDevice);

    // Kernel 1: Obtain the result of function 
    dim3 blockSize1(16, 16);
    dim3 threadperblock1((results.cols()*sigmapts.rows() + blockSize1.x - 1) / blockSize1.x, (results.rows() + blockSize1.y - 1) / blockSize1.y);

    Sigma_function<<<blockSize1, threadperblock1>>>(sigmapts_gpu, pts_gpu, mu_gpu, sigmapts.rows(), sigmapts.cols(), results.rows(), results.cols(), type, _class_gpu, _data_gpu);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(sigmapts_gpu);
    hipFree(mu_gpu);

    // Kernel 2: Obtain the result by multiplying the pts and the weights
    dim3 blockSize2(16, 16);
    dim3 threadperblock2((results.cols() + blockSize2.x - 1) / blockSize2.x, (results.rows() + blockSize2.y - 1) / blockSize2.y);

    obtain_res<<<blockSize2, threadperblock2>>>(pts_gpu, _weight_gpu, result_gpu, sigmapts.rows(), results.rows(), results.cols());
    hipDeviceSynchronize();
    hipMemcpy(results.data(), result_gpu, results.size() * sizeof(double), hipMemcpyDeviceToHost);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }

    hipFree(pts_gpu);
    hipFree(result_gpu);
}

void CudaOperation_Quad::Cuda_free()
{
    hipFree(_weight_gpu);
    hipFree(_data_gpu);
    hipFree(_class_gpu);
}


}